#include "hip/hip_runtime.h"
﻿#include "neural_radiance_caching_shared.h"

using namespace shared;

struct HitPointParameter {
    float b1, b2;
    int32_t primIndex;

    CUDA_DEVICE_FUNCTION CUDA_INLINE static HitPointParameter get() {
        HitPointParameter ret;
        float2 bc = optixGetTriangleBarycentrics();
        ret.b1 = bc.x;
        ret.b2 = bc.y;
        ret.primIndex = optixGetPrimitiveIndex();
        return ret;
    }
};

struct HitGroupSBTRecordData {
    GeometryInstanceData geomInstData;

    CUDA_DEVICE_FUNCTION CUDA_INLINE static const HitGroupSBTRecordData &get() {
        return *reinterpret_cast<HitGroupSBTRecordData*>(optixGetSbtDataPointer());
    }
};



CUDA_DEVICE_KERNEL void RT_AH_NAME(visibility)() {
    float visibility = 0.0f;
    VisibilityRayPayloadSignature::set(&visibility);
}



CUDA_DEVICE_KERNEL void RT_RG_NAME(setupGBuffers)() {
    uint2 launchIndex = make_uint2(optixGetLaunchIndex().x, optixGetLaunchIndex().y);

    const PerspectiveCamera &camera = plp.f->camera;
    float jx = 0.5f;
    float jy = 0.5f;
    if (plp.f->enableJittering) {
        PCG32RNG rng = plp.s->rngBuffer.read(launchIndex);
        jx = rng.getFloat0cTo1o();
        jy = rng.getFloat0cTo1o();
        plp.s->rngBuffer.write(launchIndex, rng);
    }
    float x = (launchIndex.x + jx) / plp.s->imageSize.x;
    float y = (launchIndex.y + jy) / plp.s->imageSize.y;
    float vh = 2 * std::tan(camera.fovY * 0.5f);
    float vw = camera.aspect * vh;

    float3 origin = camera.position;
    float3 direction = normalize(camera.orientation * make_float3(vw * (0.5f - x), vh * (0.5f - y), 1));

    HitPointParams hitPointParams;
    hitPointParams.positionInWorld = make_float3(NAN);
    hitPointParams.prevPositionInWorld = make_float3(NAN);
    hitPointParams.normalInWorld = make_float3(NAN);
    hitPointParams.texCoord = make_float2(NAN);
    hitPointParams.materialSlot = 0xFFFFFFFF;

    PickInfo pickInfo = {};

    HitPointParams* hitPointParamsPtr = &hitPointParams;
    PickInfo* pickInfoPtr = &pickInfo;
    optixu::trace<PrimaryRayPayloadSignature>(
        plp.f->travHandle, origin, direction,
        0.0f, FLT_MAX, 0.0f, 0xFF, OPTIX_RAY_FLAG_NONE,
        RayType_Primary, NumRayTypes, RayType_Primary,
        hitPointParamsPtr, pickInfoPtr);



    float2 curRasterPos = make_float2(launchIndex.x + 0.5f, launchIndex.y + 0.5f);
    float2 prevRasterPos =
        plp.f->prevCamera.calcScreenPosition(hitPointParams.prevPositionInWorld)
        * make_float2(plp.s->imageSize.x, plp.s->imageSize.y);
    float2 motionVector = curRasterPos - prevRasterPos;
    if (plp.f->resetFlowBuffer || isnan(hitPointParams.prevPositionInWorld.x))
        motionVector = make_float2(0.0f, 0.0f);

    GBuffer0 gBuffer0;
    gBuffer0.positionInWorld = hitPointParams.positionInWorld;
    gBuffer0.texCoord_x = hitPointParams.texCoord.x;
    GBuffer1 gBuffer1;
    gBuffer1.normalInWorld = hitPointParams.normalInWorld;
    gBuffer1.texCoord_y = hitPointParams.texCoord.y;
    GBuffer2 gBuffer2;
    gBuffer2.motionVector = motionVector;
    gBuffer2.materialSlot = hitPointParams.materialSlot;

    uint32_t bufIdx = plp.f->bufferIndex;
    plp.s->GBuffer0[bufIdx].write(launchIndex, gBuffer0);
    plp.s->GBuffer1[bufIdx].write(launchIndex, gBuffer1);
    plp.s->GBuffer2[bufIdx].write(launchIndex, gBuffer2);

    if (launchIndex.x == plp.f->mousePosition.x &&
        launchIndex.y == plp.f->mousePosition.y)
        *plp.f->pickInfo = pickInfo;

    // JP: デノイザーに必要な情報を出力。
    // EN: Output information required for the denoiser.
    float3 firstHitNormal = transpose(camera.orientation) * hitPointParams.normalInWorld;
    firstHitNormal.x *= -1;
    float3 prevAlbedoResult = make_float3(0.0f, 0.0f, 0.0f);
    float3 prevNormalResult = make_float3(0.0f, 0.0f, 0.0f);
    if (plp.f->numAccumFrames > 0) {
        prevAlbedoResult = getXYZ(plp.s->albedoAccumBuffer.read(launchIndex));
        prevNormalResult = getXYZ(plp.s->normalAccumBuffer.read(launchIndex));
    }
    float curWeight = 1.0f / (1 + plp.f->numAccumFrames);
    float3 albedoResult = (1 - curWeight) * prevAlbedoResult + curWeight * hitPointParams.albedo;
    float3 normalResult = (1 - curWeight) * prevNormalResult + curWeight * firstHitNormal;
    plp.s->albedoAccumBuffer.write(launchIndex, make_float4(albedoResult, 1.0f));
    plp.s->normalAccumBuffer.write(launchIndex, make_float4(normalResult, 1.0f));
}

CUDA_DEVICE_KERNEL void RT_CH_NAME(setupGBuffers)() {
    uint2 launchIndex = make_uint2(optixGetLaunchIndex().x, optixGetLaunchIndex().y);

    auto sbtr = HitGroupSBTRecordData::get();
    const InstanceData &inst = plp.f->instanceDataBuffer[optixGetInstanceId()];
    const GeometryInstanceData &geomInst = sbtr.geomInstData;

    HitPointParams* hitPointParams;
    PickInfo* pickInfo;
    PrimaryRayPayloadSignature::get(&hitPointParams, &pickInfo);

    auto hp = HitPointParameter::get();
    float3 positionInWorld;
    float3 prevPositionInWorld;
    float3 shadingNormalInWorld;
    float3 texCoord0DirInWorld;
    //float3 geometricNormalInWorld;
    float2 texCoord;
    {
        const Triangle &tri = geomInst.triangleBuffer[hp.primIndex];
        const Vertex &v0 = geomInst.vertexBuffer[tri.index0];
        const Vertex &v1 = geomInst.vertexBuffer[tri.index1];
        const Vertex &v2 = geomInst.vertexBuffer[tri.index2];
        float b1 = hp.b1;
        float b2 = hp.b2;
        float b0 = 1 - (b1 + b2);
        float3 localP = b0 * v0.position + b1 * v1.position + b2 * v2.position;
        shadingNormalInWorld = b0 * v0.normal + b1 * v1.normal + b2 * v2.normal;
        texCoord0DirInWorld = b0 * v0.texCoord0Dir + b1 * v1.texCoord0Dir + b2 * v2.texCoord0Dir;
        //geometricNormalInWorld = cross(v1.position - v0.position, v2.position - v0.position);
        texCoord = b0 * v0.texCoord + b1 * v1.texCoord + b2 * v2.texCoord;

        positionInWorld = optixTransformPointFromObjectToWorldSpace(localP);
        prevPositionInWorld = inst.prevTransform * localP;
        shadingNormalInWorld = normalize(optixTransformNormalFromObjectToWorldSpace(shadingNormalInWorld));
        texCoord0DirInWorld = normalize(optixTransformVectorFromObjectToWorldSpace(texCoord0DirInWorld));
        //geometricNormalInWorld = normalize(optixTransformNormalFromObjectToWorldSpace(geometricNormalInWorld));
        if (!allFinite(shadingNormalInWorld)) {
            shadingNormalInWorld = make_float3(0, 0, 1);
            texCoord0DirInWorld = make_float3(1, 0, 0);
        }
    }

    const MaterialData &mat = plp.s->materialDataBuffer[geomInst.materialSlot];

    BSDF bsdf;
    bsdf.setup(mat, texCoord);
    ReferenceFrame shadingFrame(shadingNormalInWorld, texCoord0DirInWorld);
    float3 modLocalNormal = mat.readModifiedNormal(mat.normal, texCoord, mat.normalDimension);
    if (plp.f->enableBumpMapping)
        applyBumpMapping(modLocalNormal, &shadingFrame);
    float3 vOut = -optixGetWorldRayDirection();
    float3 vOutLocal = shadingFrame.toLocal(normalize(vOut));

    hitPointParams->albedo = bsdf.evaluateDHReflectanceEstimate(vOutLocal);
    hitPointParams->positionInWorld = positionInWorld;
    hitPointParams->prevPositionInWorld = prevPositionInWorld;
    hitPointParams->normalInWorld = shadingFrame.normal;
    hitPointParams->texCoord = texCoord;
    hitPointParams->materialSlot = geomInst.materialSlot;

    // JP: マウスが乗っているピクセルの情報を出力する。
    // EN: Export the information of the pixel on which the mouse is.
    if (launchIndex.x == plp.f->mousePosition.x &&
        launchIndex.y == plp.f->mousePosition.y) {
        pickInfo->hit = true;
        pickInfo->instSlot = optixGetInstanceId();
        pickInfo->geomInstSlot = geomInst.geomInstSlot;
        pickInfo->matSlot = geomInst.materialSlot;
        pickInfo->primIndex = hp.primIndex;
        pickInfo->positionInWorld = positionInWorld;
        pickInfo->normalInWorld = shadingFrame.normal;
        pickInfo->albedo = hitPointParams->albedo;
        float3 emittance = make_float3(0.0f, 0.0f, 0.0f);
        if (mat.emittance) {
            float4 texValue = tex2DLod<float4>(mat.emittance, texCoord.x, texCoord.y, 0.0f);
            emittance = make_float3(texValue);
        }
        pickInfo->emittance = emittance;
    }
}

CUDA_DEVICE_KERNEL void RT_MS_NAME(setupGBuffers)() {
    uint2 launchIndex = make_uint2(optixGetLaunchIndex().x, optixGetLaunchIndex().y);

    float3 vOut = -optixGetWorldRayDirection();
    float3 p = -vOut;

    float posPhi, posTheta;
    toPolarYUp(p, &posPhi, &posTheta);

    float phi = posPhi + plp.f->envLightRotation;

    float u = phi / (2 * Pi);
    u -= floorf(u);
    float v = posTheta / Pi;

    HitPointParams* hitPointParams;
    PickInfo* pickInfo;
    PrimaryRayPayloadSignature::get(&hitPointParams, &pickInfo);

    hitPointParams->albedo = make_float3(0.0f, 0.0f, 0.0f);
    hitPointParams->positionInWorld = p;
    hitPointParams->prevPositionInWorld = p;
    hitPointParams->normalInWorld = vOut;
    hitPointParams->texCoord = make_float2(u, v);
    hitPointParams->materialSlot = 0xFFFFFFFF;

    // JP: マウスが乗っているピクセルの情報を出力する。
    // EN: Export the information of the pixel on which the mouse is.
    if (launchIndex.x == plp.f->mousePosition.x &&
        launchIndex.y == plp.f->mousePosition.y) {
        pickInfo->hit = true;
        pickInfo->instSlot = 0xFFFFFFFF;
        pickInfo->geomInstSlot = 0xFFFFFFFF;
        pickInfo->matSlot = 0xFFFFFFFF;
        pickInfo->primIndex = 0xFFFFFFFF;
        pickInfo->positionInWorld = p;
        pickInfo->albedo = make_float3(0.0f, 0.0f, 0.0f);
        float3 emittance = make_float3(0.0f, 0.0f, 0.0f);
        if (plp.s->envLightTexture && plp.f->enableEnvLight) {
            float4 texValue = tex2DLod<float4>(plp.s->envLightTexture, u, v, 0.0f);
            emittance = make_float3(texValue);
            emittance *= Pi * plp.f->envLightPowerCoeff;
        }
        pickInfo->emittance = emittance;
        pickInfo->normalInWorld = vOut;
    }
}



CUDA_DEVICE_FUNCTION CUDA_INLINE void convertToPolar(const float3 &dir, float* phi, float* theta) {
    float z = std::fmin(std::fmax(dir.z, -1.0f), 1.0f);
    *theta = std::acos(z);
    *phi = std::atan2(dir.y, dir.x);
}

CUDA_DEVICE_FUNCTION CUDA_INLINE void createRadianceQuery(
    const float3 &positionInWorld, const float3 &normalInWorld, const float3 &scatteredDirInWorld,
    float roughness, const float3 &diffuseReflectance, const float3 &specularReflectance,
    RadianceQuery* query) {
    float phi, theta;
    query->position = plp.s->sceneAABB->normalize(positionInWorld);
    convertToPolar(normalInWorld, &phi, &theta);
    query->normal_phi = phi;
    query->normal_theta = theta;
    convertToPolar(scatteredDirInWorld, &phi, &theta);
    query->vOut_phi = phi;
    query->vOut_theta = theta;
    query->roughness = roughness;
    query->diffuseReflectance = diffuseReflectance;
    query->specularReflectance = specularReflectance;
}

static constexpr bool useSolidAngleSampling = false;

CUDA_DEVICE_FUNCTION CUDA_INLINE float3 performNextEventEstimation(
    const float3 &shadingPoint, const float3 &vOutLocal, const ReferenceFrame &shadingFrame, const BSDF &bsdf,
    PCG32RNG &rng) {
    float uLight = rng.getFloat0cTo1o();
    bool selectEnvLight = false;
    float probToSampleCurLightType = 1.0f;
    if (plp.s->envLightTexture && plp.f->enableEnvLight) {
        if (uLight < probToSampleEnvLight) {
            probToSampleCurLightType = probToSampleEnvLight;
            uLight /= probToSampleCurLightType;
            selectEnvLight = true;
        }
        else {
            probToSampleCurLightType = 1.0f - probToSampleEnvLight;
            uLight = (uLight - probToSampleEnvLight) / probToSampleCurLightType;
        }
    }
    LightSample lightSample;
    float areaPDensity;
    sampleLight<useSolidAngleSampling>(
        shadingPoint,
        uLight, selectEnvLight, rng.getFloat0cTo1o(), rng.getFloat0cTo1o(),
        &lightSample, &areaPDensity);
    areaPDensity *= probToSampleCurLightType;

    float3 shadowRay = lightSample.atInfinity ?
        lightSample.position :
        (lightSample.position - shadingPoint);
    float dist2 = sqLength(shadowRay);
    shadowRay /= std::sqrt(dist2);
    float3 vInLocal = shadingFrame.toLocal(shadowRay);
    float lpCos = std::fabs(dot(shadowRay, lightSample.normal));
    float bsdfPDensity = bsdf.evaluatePDF(vOutLocal, vInLocal) * lpCos / dist2;
    if (!isfinite(bsdfPDensity))
        bsdfPDensity = 0.0f;
    float lightPDensity = areaPDensity;
    float misWeight = pow2(lightPDensity) / (pow2(bsdfPDensity) + pow2(lightPDensity));
    float3 ret = make_float3(0.0f);
    if (areaPDensity > 0.0f)
        ret = performDirectLighting<true>(
            shadingPoint, vOutLocal, shadingFrame, bsdf, lightSample) * (misWeight / areaPDensity);

    return ret;
}

template <bool useNRC>
CUDA_DEVICE_FUNCTION CUDA_INLINE void pathTrace_raygen_generic() {
    uint2 launchIndex = make_uint2(optixGetLaunchIndex().x, optixGetLaunchIndex().y);

    uint32_t bufIdx = plp.f->bufferIndex;
    GBuffer0 gBuffer0 = plp.s->GBuffer0[bufIdx].read(launchIndex);
    GBuffer1 gBuffer1 = plp.s->GBuffer1[bufIdx].read(launchIndex);
    GBuffer2 gBuffer2 = plp.s->GBuffer2[bufIdx].read(launchIndex);

    float3 positionInWorld = gBuffer0.positionInWorld;
    float3 shadingNormalInWorld = gBuffer1.normalInWorld;
    float2 texCoord = make_float2(gBuffer0.texCoord_x, gBuffer1.texCoord_y);
    uint32_t materialSlot = gBuffer2.materialSlot;

    const PerspectiveCamera &camera = plp.f->camera;

    uint32_t linearTileIndex;
    bool isTrainingPath;
    bool isUnbiasedTrainingTile;
    if constexpr (useNRC) {
        const uint2 tileSize = *plp.s->tileSize[bufIdx];
        const uint32_t numPixelsInTile = tileSize.x * tileSize.y;

        // JP: 動的サイズのタイルごとに1つトレーニングパスを選ぶ。
        // EN: choose a training path for each dynamic-sized tile.
        uint2 localIndex = launchIndex % tileSize;
        uint32_t localLinearIndex = localIndex.y * tileSize.x + localIndex.x;
        isTrainingPath = (localLinearIndex + *plp.s->offsetToSelectTrainingPath) % numPixelsInTile == 0;

        uint2 numTiles = (plp.s->imageSize + tileSize - 1) / tileSize;
        uint2 tileIndex = launchIndex / tileSize;
        linearTileIndex = tileIndex.y * numTiles.x + tileIndex.x;

        // JP: トレーニングパスの16本に1本はセルフトレーニングを使用しないUnbiasedパスとする。
        // EN: Make one path out of every 16 training paths not use self-training and unbiased.
        const uint2 tileGroupSize = make_uint2(4, 4);
        uint2 localTileIndex = tileIndex % tileGroupSize;
        uint32_t localLinearTileIndex = localTileIndex.y * tileGroupSize.x + localTileIndex.x;
        isUnbiasedTrainingTile = (localLinearTileIndex + *plp.s->offsetToSelectUnbiasedTile) % 16 == 0;
    }
    else {
        (void)linearTileIndex;
        (void)isTrainingPath;
        (void)isUnbiasedTrainingTile;
    }

    bool useEnvLight = plp.s->envLightTexture && plp.f->enableEnvLight;
    float3 contribution = make_float3(0.001f, 0.001f, 0.001f);
    bool renderingPathEndsWithCache = false;
    uint32_t pathLength = 1;
    if (materialSlot != 0xFFFFFFFF) {
        float3 alpha = make_float3(1.0f);
        float initImportance = sRGB_calcLuminance(alpha);
        PCG32RNG rng = plp.s->rngBuffer.read(launchIndex);

        // JP: 最初の交点におけるシェーディング。
        // EN: Shading on the first hit.
        float3 vIn;
        float dirPDensity;
        float primaryPathSpread;
        float3 localThroughput;
        uint32_t trainDataIndex;
        {
            const MaterialData &mat = plp.s->materialDataBuffer[materialSlot];

            // TODO?: Use true geometric normal.
            float3 geometricNormalInWorld = shadingNormalInWorld;
            float3 vOut = camera.position - positionInWorld;
            float primaryDist2 = sqLength(vOut);
            vOut /= std::sqrt(primaryDist2);
            float primaryDotVN = dot(vOut, geometricNormalInWorld);
            float frontHit = primaryDotVN >= 0.0f ? 1.0f : -1.0f;

            if constexpr (useNRC)
                primaryPathSpread = primaryDist2 / (4 * Pi * std::fabs(primaryDotVN));

            ReferenceFrame shadingFrame(shadingNormalInWorld);
            positionInWorld = offsetRayOriginNaive(positionInWorld, frontHit * geometricNormalInWorld);
            float3 vOutLocal = shadingFrame.toLocal(vOut);

            // JP: 光源を直接見ている場合の寄与を蓄積。
            // EN: Accumulate the contribution from a light source directly seeing.
            contribution = make_float3(0.0f);
            if (vOutLocal.z > 0 && mat.emittance) {
                float4 texValue = tex2DLod<float4>(mat.emittance, texCoord.x, texCoord.y, 0.0f);
                float3 emittance = make_float3(texValue);
                contribution += alpha * emittance / Pi;
            }

            BSDF bsdf;
            bsdf.setup(mat, texCoord);

            // Next event estimation (explicit light sampling) on the first hit.
            float3 directContNEE = performNextEventEstimation(
                positionInWorld, vOutLocal, shadingFrame, bsdf, rng);
            contribution += alpha * directContNEE;

            // generate a next ray.
            float3 vInLocal;
            localThroughput = bsdf.sampleThroughput(
                vOutLocal, rng.getFloat0cTo1o(), rng.getFloat0cTo1o(),
                &vInLocal, &dirPDensity);
            alpha *= localThroughput;
            vIn = shadingFrame.fromLocal(vInLocal);

            if constexpr (useNRC) {
                // JP: 訓練データエントリーの確保。
                // EN: Allocate a training data entry.
                if (isTrainingPath) {
                    trainDataIndex = atomicAdd(plp.s->numTrainingData[bufIdx], 1u);

                    if (trainDataIndex < trainBufferSize) {
                        float roughness;
                        float3 diffuseReflectance, specularReflectance;
                        bsdf.getSurfaceParameters(
                            &diffuseReflectance, &specularReflectance, &roughness);

                        RadianceQuery radQuery;
                        createRadianceQuery(
                            positionInWorld, shadingFrame.normal, vOut,
                            roughness, diffuseReflectance, specularReflectance,
                            &radQuery);
                        plp.s->trainRadianceQueryBuffer[0][trainDataIndex] = radQuery;

                        TrainingVertexInfo vertInfo;
                        vertInfo.localThroughput = localThroughput;
                        vertInfo.prevVertexDataIndex = invalidVertexDataIndex;
                        vertInfo.pathLength = pathLength;
                        plp.s->trainVertexInfoBuffer[trainDataIndex] = vertInfo;

                        // JP: 現在の頂点に対する直接照明(NEE)によるScattered Radianceでターゲット値を初期化。
                        // EN: Initialize a target value by scattered radiance at the current vertex
                        //     by direct lighting (NEE).
                        plp.s->trainTargetBuffer[0][trainDataIndex] = directContNEE;
                        //if (!allFinite(directContNEE))
                        //    printf("NEE: (%g, %g, %g)\n",
                        //           directContNEE.x, directContNEE.y, directContNEE.z);
                    }
                    else {
                        trainDataIndex = invalidVertexDataIndex;
                    }
                }
            }
            else {
                (void)primaryPathSpread;
                (void)trainDataIndex;
            }
        }

        // Path extension loop
        PathTraceWriteOnlyPayload woPayload = {};
        PathTraceWriteOnlyPayload* woPayloadPtr = &woPayload;
        PathTraceReadWritePayload<useNRC> rwPayload = {};
        PathTraceReadWritePayload<useNRC>* rwPayloadPtr = &rwPayload;
        rwPayload.rng = rng;
        rwPayload.initImportance = initImportance;
        rwPayload.alpha = alpha;
        rwPayload.contribution = contribution;
        rwPayload.prevDirPDensity = dirPDensity;
        if constexpr (useNRC) {
            rwPayload.linearTileIndex = linearTileIndex;
            rwPayload.primaryPathSpread = primaryPathSpread;
            rwPayload.curSqrtPathSpread = 0.0f;
            rwPayload.prevLocalThroughput = localThroughput;
            rwPayload.prevTrainDataIndex = trainDataIndex;
            rwPayload.renderingPathEndsWithCache = false;
            rwPayload.isTrainingPath = isTrainingPath;
            rwPayload.isUnbiasedTrainingTile = isUnbiasedTrainingTile;
            rwPayload.trainingSuffixEndsWithCache = false;
        }
        rwPayload.pathLength = pathLength;
        float3 rayOrg = positionInWorld;
        float3 rayDir = vIn;
        while (true) {
            bool isValidSampling = rwPayload.prevDirPDensity > 0.0f && isfinite(rwPayload.prevDirPDensity);
            if (!isValidSampling)
                break;

            ++rwPayload.pathLength;
            // JP: 通常のパストレーシングとNRCを正しく比較するには(特に通常のパストレーシングにおいて)
            //     反射回数制限を解除する必要がある。
            // EN: Disabling the limitation in the number of bounces (particularly for the base path tracing)
            //     is required to properly compare the base path tracing and NRC.
            if (rwPayload.pathLength >= plp.f->maxPathLength)
                rwPayload.maxLengthTerminate = true;
            rwPayload.terminate = true;

            constexpr RayType pathTraceRayType = useNRC ? RayType_PathTraceNRC : RayType_PathTraceBaseline;
            optixu::trace<PathTraceRayPayloadSignature<useNRC>>(
                plp.f->travHandle, rayOrg, rayDir,
                0.0f, FLT_MAX, 0.0f, 0xFF, OPTIX_RAY_FLAG_NONE,
                pathTraceRayType, NumRayTypes, pathTraceRayType,
                woPayloadPtr, rwPayloadPtr);
            if (rwPayload.terminate)
                break;
            rayOrg = woPayload.nextOrigin;
            rayDir = woPayload.nextDirection;
        }
        contribution = rwPayload.contribution;

        plp.s->rngBuffer.write(launchIndex, rwPayload.rng);

        if constexpr (useNRC) {
            renderingPathEndsWithCache = rwPayload.renderingPathEndsWithCache;
            pathLength = rwPayload.pathLength;
            if (rwPayload.isTrainingPath && !rwPayload.trainingSuffixEndsWithCache) {
                TrainingSuffixTerminalInfo terminalInfo;
                terminalInfo.prevVertexDataIndex = rwPayload.prevTrainDataIndex;
                terminalInfo.hasQuery = false;
                terminalInfo.pathLength = rwPayload.pathLength;
                plp.s->trainSuffixTerminalInfoBuffer[rwPayload.linearTileIndex] = terminalInfo;
            }
        }
    }
    else {
        // JP: 環境光源を直接見ている場合の寄与を蓄積。
        // EN: Accumulate the contribution from the environmental light source directly seeing.
        if (useEnvLight) {
            float u = texCoord.x, v = texCoord.y;
            float4 texValue = tex2DLod<float4>(plp.s->envLightTexture, u, v, 0.0f);
            float3 luminance = plp.f->envLightPowerCoeff * make_float3(texValue);
            contribution = luminance;
        }
    }

    if constexpr (useNRC) {
        uint32_t linearIndex = launchIndex.y * plp.s->imageSize.x + launchIndex.x;

        // JP: 無限遠にレイが飛んだか、ロシアンルーレットによってパストレースが完了したケース。
        // EN: When a ray goes infinity or the path ends with Russain roulette.
        if (!renderingPathEndsWithCache) {
            TerminalInfo terminalInfo;
            terminalInfo.alpha = make_float3(0.0f, 0.0f, 0.0f);
            terminalInfo.pathLength = pathLength;
            terminalInfo.hasQuery = false;
            terminalInfo.isTrainingPixel = isTrainingPath;
            terminalInfo.isUnbiasedTile = isUnbiasedTrainingTile;
            plp.s->inferenceTerminalInfoBuffer[linearIndex] = terminalInfo;
        }

        plp.s->perFrameContributionBuffer[linearIndex] = contribution;
    }
    else {
        (void)renderingPathEndsWithCache;
        (void)pathLength;

        float3 prevColorResult = make_float3(0.0f, 0.0f, 0.0f);
        if (plp.f->numAccumFrames > 0)
            prevColorResult = getXYZ(plp.s->beautyAccumBuffer.read(launchIndex));
        float curWeight = 1.0f / (1 + plp.f->numAccumFrames);
        float3 colorResult = (1 - curWeight) * prevColorResult + curWeight * contribution;
        plp.s->beautyAccumBuffer.write(launchIndex, make_float4(colorResult, 1.0f));
    }
}

template <bool useNRC>
CUDA_DEVICE_FUNCTION CUDA_INLINE void pathTrace_closestHit_generic() {
    uint2 launchIndex = make_uint2(optixGetLaunchIndex().x, optixGetLaunchIndex().y);
    uint32_t bufIdx = plp.f->bufferIndex;

    auto sbtr = HitGroupSBTRecordData::get();
    const InstanceData &inst = plp.f->instanceDataBuffer[optixGetInstanceId()];
    const GeometryInstanceData &geomInst = sbtr.geomInstData;

    PathTraceWriteOnlyPayload* woPayload;
    PathTraceReadWritePayload<useNRC>* rwPayload;
    PathTraceRayPayloadSignature<useNRC>::get(&woPayload, &rwPayload);
    PCG32RNG &rng = rwPayload->rng;

    const float3 rayOrigin = optixGetWorldRayOrigin();

    auto hp = HitPointParameter::get();
    float3 positionInWorld;
    float3 shadingNormalInWorld;
    float3 texCoord0DirInWorld;
    float3 geometricNormalInWorld;
    float2 texCoord;
    float hypAreaPDensity;
    computeSurfacePoint<true, useSolidAngleSampling>(
        inst, geomInst, hp.primIndex, hp.b1, hp.b2,
        rayOrigin,
        &positionInWorld, &shadingNormalInWorld, &texCoord0DirInWorld,
        &geometricNormalInWorld, &texCoord, &hypAreaPDensity);

    const MaterialData &mat = plp.s->materialDataBuffer[geomInst.materialSlot];

    float3 vOut = normalize(-optixGetWorldRayDirection());
    float frontHit = dot(vOut, geometricNormalInWorld) >= 0.0f ? 1.0f : -1.0f;

    ReferenceFrame shadingFrame(shadingNormalInWorld, texCoord0DirInWorld);
    float3 modLocalNormal = mat.readModifiedNormal(mat.normal, texCoord, mat.normalDimension);
    if (plp.f->enableBumpMapping)
        applyBumpMapping(modLocalNormal, &shadingFrame);
    positionInWorld = offsetRayOrigin(positionInWorld, frontHit * geometricNormalInWorld);
    float3 vOutLocal = shadingFrame.toLocal(vOut);

    float dist2 = squaredDistance(rayOrigin, positionInWorld);
    if constexpr (useNRC)
        rwPayload->curSqrtPathSpread += std::sqrt(dist2 / (rwPayload->prevDirPDensity * std::fabs(vOutLocal.z)));

    // Implicit Light Sampling
    if (vOutLocal.z > 0 && mat.emittance) {
        float4 texValue = tex2DLod<float4>(mat.emittance, texCoord.x, texCoord.y, 0.0f);
        float3 emittance = make_float3(texValue);
        float lightPDensity = hypAreaPDensity * dist2 / vOutLocal.z;
        float bsdfPDensity = rwPayload->prevDirPDensity;
        float misWeight = pow2(bsdfPDensity) / (pow2(bsdfPDensity) + pow2(lightPDensity));
        float3 directContImplicit = emittance * (misWeight / Pi);
        rwPayload->contribution += rwPayload->alpha * directContImplicit;

        if constexpr (useNRC) {
            // JP: 1つ前の頂点に対する直接照明(Implicit)によるScattered Radianceをターゲット値に加算。
            // EN: Accumulate scattered radiance at the previous vertex by direct lighting (implicit)
            //     to the target value.
            if (rwPayload->isTrainingPath && rwPayload->prevTrainDataIndex != invalidVertexDataIndex) {
                plp.s->trainTargetBuffer[0][rwPayload->prevTrainDataIndex] +=
                    rwPayload->prevLocalThroughput * directContImplicit;
                //if (!allFinite(rwPayload->prevLocalThroughput) ||
                //    !allFinite(directContImplicit))
                //    printf("Implicit: (%g, %g, %g), (%g, %g, %g)\n",
                //           rwPayload->prevLocalThroughput.x,
                //           rwPayload->prevLocalThroughput.y,
                //           rwPayload->prevLocalThroughput.z,
                //           directContImplicit.x,
                //           directContImplicit.y,
                //           directContImplicit.z);
            }
        }
    }

    // Russian roulette
    bool performRR = true;
    if constexpr (useNRC) {
        if (rwPayload->isTrainingPath)
            performRR = rwPayload->pathLength > 2;
    }
    if (performRR) {
        float continueProb = std::fmin(sRGB_calcLuminance(rwPayload->alpha) / rwPayload->initImportance, 1.0f);
        if (rng.getFloat0cTo1o() >= continueProb || rwPayload->maxLengthTerminate)
            return;
        float recContinueProb = 1.0f / continueProb;
        rwPayload->alpha *= recContinueProb;
        if constexpr (useNRC) {
            if (rwPayload->isTrainingPath && rwPayload->prevTrainDataIndex != invalidVertexDataIndex)
                plp.s->trainVertexInfoBuffer[rwPayload->prevTrainDataIndex].localThroughput *= recContinueProb;
        }
    }

    BSDF bsdf;
    bsdf.setup(mat, texCoord);

    if constexpr (useNRC) {
        bool endsWithCache = false;
        bool pathIsSpreadEnough =
            pow2(rwPayload->curSqrtPathSpread) > pathTerminationFactor * rwPayload->primaryPathSpread;
        endsWithCache |= pathIsSpreadEnough;
        if (rwPayload->renderingPathEndsWithCache &&
            rwPayload->isTrainingPath && rwPayload->isUnbiasedTrainingTile)
            endsWithCache = false;

        if (endsWithCache) {
            uint32_t linearIndex = launchIndex.y * plp.s->imageSize.x + launchIndex.x;

            float roughness;
            float3 diffuseReflectance, specularReflectance;
            bsdf.getSurfaceParameters(
                &diffuseReflectance, &specularReflectance, &roughness);

            // JP: Radianceクエリーのための情報を記録する。
            // EN: Store information for radiance query.
            RadianceQuery radQuery;
            createRadianceQuery(
                positionInWorld, shadingFrame.normal, vOut,
                roughness, diffuseReflectance, specularReflectance,
                &radQuery);

            if (!rwPayload->renderingPathEndsWithCache) {
                plp.s->inferenceRadianceQueryBuffer[linearIndex] = radQuery;

                TerminalInfo terminalInfo;
                terminalInfo.alpha = rwPayload->alpha;
                terminalInfo.pathLength = rwPayload->pathLength;
                terminalInfo.hasQuery = true;
                terminalInfo.isTrainingPixel = rwPayload->isTrainingPath;
                terminalInfo.isUnbiasedTile = rwPayload->isUnbiasedTrainingTile;
                plp.s->inferenceTerminalInfoBuffer[linearIndex] = terminalInfo;

                rwPayload->renderingPathEndsWithCache = true;
                if (rwPayload->isTrainingPath)
                    rwPayload->curSqrtPathSpread = 0;
                else
                    return;
            }
            else {
                // JP: 訓練データバッファーがフルの場合は既にTraining Suffixは終了したことになっている。
                // EN: The training suffix should have been ended if the training data buffer is full.
                if (!rwPayload->trainingSuffixEndsWithCache) {
                    uint32_t offset = plp.s->imageSize.x * plp.s->imageSize.y;
                    plp.s->inferenceRadianceQueryBuffer[offset + rwPayload->linearTileIndex] = radQuery;

                    // JP: 直前のTraining VertexへのリンクとともにTraining Suffixを終了させる。
                    // EN: Finish the training suffix with the link to the previous training vertex.
                    TrainingSuffixTerminalInfo terminalInfo;
                    terminalInfo.prevVertexDataIndex = rwPayload->prevTrainDataIndex;
                    terminalInfo.hasQuery = true;
                    terminalInfo.pathLength = rwPayload->pathLength;
                    plp.s->trainSuffixTerminalInfoBuffer[rwPayload->linearTileIndex] = terminalInfo;

                    rwPayload->trainingSuffixEndsWithCache = true;
                }
                return;
            }
        }
    }

    // Next Event Estimation (Explicit Light Sampling)
    float3 directContNEE = performNextEventEstimation(
        positionInWorld, vOutLocal, shadingFrame, bsdf, rng);
    rwPayload->contribution += rwPayload->alpha * directContNEE;

    // generate a next ray.
    float3 vInLocal;
    float dirPDensity;
    float3 localThroughput = bsdf.sampleThroughput(
        vOutLocal, rng.getFloat0cTo1o(), rng.getFloat0cTo1o(),
        &vInLocal, &dirPDensity);
    rwPayload->alpha *= localThroughput;
    float3 vIn = shadingFrame.fromLocal(vInLocal);

    woPayload->nextOrigin = positionInWorld;
    woPayload->nextDirection = vIn;
    rwPayload->prevDirPDensity = dirPDensity;
    if constexpr (useNRC)
        rwPayload->prevLocalThroughput = localThroughput;
    rwPayload->terminate = false;

    if constexpr (useNRC) {
        // JP: 訓練データエントリーの確保。
        // EN: Allocate a training data entry.
        if (rwPayload->isTrainingPath && !rwPayload->trainingSuffixEndsWithCache) {
            uint32_t trainDataIndex = atomicAdd(plp.s->numTrainingData[bufIdx], 1u);

            // TODO?: 訓練データ数の正確な推定のためにtrainingSuffixEndsWithCacheのチェックをここに持ってくる？

            float roughness;
            float3 diffuseReflectance, specularReflectance;
            bsdf.getSurfaceParameters(
                &diffuseReflectance, &specularReflectance, &roughness);

            RadianceQuery radQuery;
            createRadianceQuery(
                positionInWorld, shadingFrame.normal, vOut,
                roughness, diffuseReflectance, specularReflectance,
                &radQuery);

            if (trainDataIndex < trainBufferSize) {
                plp.s->trainRadianceQueryBuffer[0][trainDataIndex] = radQuery;

                // JP: ローカルスループットと前のTraining Vertexへのリンクを記録。
                // EN: Record the local throughput and the link to the previous training vertex.
                TrainingVertexInfo vertInfo;
                vertInfo.localThroughput = localThroughput;
                vertInfo.prevVertexDataIndex = rwPayload->prevTrainDataIndex;
                vertInfo.pathLength = rwPayload->pathLength;
                plp.s->trainVertexInfoBuffer[trainDataIndex] = vertInfo;

                // JP: 現在の頂点に対する直接照明(NEE)によるScattered Radianceでターゲット値を初期化。
                // EN: Initialize a target value by scattered radiance at the current vertex by
                //     direct lighting (NEE).
                plp.s->trainTargetBuffer[0][trainDataIndex] = directContNEE;
                //if (!allFinite(directContNEE))
                //    printf("NEE: (%g, %g, %g)\n",
                //           directContNEE.x, directContNEE.y, directContNEE.z);

                rwPayload->prevTrainDataIndex = trainDataIndex;
            }
            // JP: 訓練データがバッファーを溢れた場合は強制的にTraining Suffixを終了させる。
            // EN: Forcefully end the training suffix if the training data buffer become full.
            else {
                uint32_t offset = plp.s->imageSize.x * plp.s->imageSize.y;
                plp.s->inferenceRadianceQueryBuffer[offset + rwPayload->linearTileIndex] = radQuery;

                TrainingSuffixTerminalInfo terminalInfo;
                terminalInfo.prevVertexDataIndex = rwPayload->prevTrainDataIndex;
                terminalInfo.hasQuery = true;
                terminalInfo.pathLength = rwPayload->pathLength;
                plp.s->trainSuffixTerminalInfoBuffer[rwPayload->linearTileIndex] = terminalInfo;

                rwPayload->trainingSuffixEndsWithCache = true;
            }
        }
    }
}

template <bool useNRC>
CUDA_DEVICE_FUNCTION CUDA_INLINE void pathTrace_miss_generic() {
    if (!plp.s->envLightTexture || !plp.f->enableEnvLight)
        return;

    PathTraceReadWritePayload<useNRC>* rwPayload;
    PathTraceRayPayloadSignature<useNRC>::get(nullptr, &rwPayload);

    float3 rayDir = normalize(optixGetWorldRayDirection());
    float posPhi, theta;
    toPolarYUp(rayDir, &posPhi, &theta);

    float phi = posPhi + plp.f->envLightRotation;
    phi = phi - floorf(phi / (2 * Pi)) * 2 * Pi;
    float2 texCoord = make_float2(phi / (2 * Pi), theta / Pi);

    // Implicit Light Sampling
    float4 texValue = tex2DLod<float4>(plp.s->envLightTexture, texCoord.x, texCoord.y, 0.0f);
    float3 luminance = plp.f->envLightPowerCoeff * make_float3(texValue);
    float uvPDF = plp.s->envLightImportanceMap.evaluatePDF(texCoord.x, texCoord.y);
    float hypAreaPDensity = uvPDF / (2 * Pi * Pi * std::sin(theta));
    float lightPDensity = probToSampleEnvLight * hypAreaPDensity;
    float bsdfPDensity = rwPayload->prevDirPDensity;
    float misWeight = pow2(bsdfPDensity) / (pow2(bsdfPDensity) + pow2(lightPDensity));
    float3 directContImplicit = misWeight * luminance;
    rwPayload->contribution += rwPayload->alpha * directContImplicit;

    if constexpr (useNRC) {
        // JP: 1つ前の頂点に対する直接照明(Implicit)によるScattered Radianceをターゲット値に加算。
        // EN: Accumulate scattered radiance at the previous vertex by direct lighting (implicit)
        //     to the target value.
        if (rwPayload->isTrainingPath) {
            plp.s->trainTargetBuffer[0][rwPayload->prevTrainDataIndex] +=
                rwPayload->prevLocalThroughput * directContImplicit;
        }
    }
}



CUDA_DEVICE_KERNEL void RT_RG_NAME(pathTraceBaseline)() {
    pathTrace_raygen_generic<false>();
}

CUDA_DEVICE_KERNEL void RT_CH_NAME(pathTraceBaseline)() {
    pathTrace_closestHit_generic<false>();
}

CUDA_DEVICE_KERNEL void RT_MS_NAME(pathTraceBaseline)() {
    pathTrace_miss_generic<false>();
}



CUDA_DEVICE_KERNEL void RT_RG_NAME(pathTraceNRC)() {
    pathTrace_raygen_generic<true>();
}

CUDA_DEVICE_KERNEL void RT_CH_NAME(pathTraceNRC)() {
    pathTrace_closestHit_generic<true>();
}

CUDA_DEVICE_KERNEL void RT_MS_NAME(pathTraceNRC)() {
    pathTrace_miss_generic<true>();
}



CUDA_DEVICE_KERNEL void RT_RG_NAME(visualizePrediction)() {
    uint2 launchIndex = make_uint2(optixGetLaunchIndex().x, optixGetLaunchIndex().y);
    uint32_t linearIndex = launchIndex.y * plp.s->imageSize.x + launchIndex.x;

    uint32_t bufIdx = plp.f->bufferIndex;
    GBuffer0 gBuffer0 = plp.s->GBuffer0[bufIdx].read(launchIndex);
    GBuffer1 gBuffer1 = plp.s->GBuffer1[bufIdx].read(launchIndex);
    GBuffer2 gBuffer2 = plp.s->GBuffer2[bufIdx].read(launchIndex);

    float3 positionInWorld = gBuffer0.positionInWorld;
    float3 shadingNormalInWorld = gBuffer1.normalInWorld;
    float2 texCoord = make_float2(gBuffer0.texCoord_x, gBuffer1.texCoord_y);
    uint32_t materialSlot = gBuffer2.materialSlot;

    const PerspectiveCamera &camera = plp.f->camera;

    if (materialSlot != 0xFFFFFFFF) {
        const MaterialData &mat = plp.s->materialDataBuffer[materialSlot];

        // TODO?: Use true geometric normal.
        float3 geometricNormalInWorld = shadingNormalInWorld;
        float3 vOut = camera.position - positionInWorld;
        float primaryDist2 = sqLength(vOut);
        vOut /= std::sqrt(primaryDist2);
        float primaryDotVN = dot(vOut, geometricNormalInWorld);
        float frontHit = primaryDotVN >= 0.0f ? 1.0f : -1.0f;

        ReferenceFrame shadingFrame(shadingNormalInWorld);
        positionInWorld = offsetRayOriginNaive(positionInWorld, frontHit * geometricNormalInWorld);

        BSDF bsdf;
        bsdf.setup(mat, texCoord);

        float roughness;
        float3 diffuseReflectance, specularReflectance;
        bsdf.getSurfaceParameters(
            &diffuseReflectance, &specularReflectance, &roughness);

        RadianceQuery radQuery;
        createRadianceQuery(
            positionInWorld, shadingFrame.normal, vOut,
            roughness, diffuseReflectance, specularReflectance,
            &radQuery);

        plp.s->inferenceRadianceQueryBuffer[linearIndex] = radQuery;
    }
    else {
        //// JP: 環境光源を直接見ている場合の寄与を蓄積。
        //// EN: Accumulate the contribution from the environmental light source directly seeing.
        //if (useEnvLight) {
        //    float u = texCoord.x, v = texCoord.y;
        //    float4 texValue = tex2DLod<float4>(plp.s->envLightTexture, u, v, 0.0f);
        //    float3 luminance = plp.f->envLightPowerCoeff * make_float3(texValue);
        //    contribution = luminance;
        //}
    }

    TerminalInfo terminalInfo;
    terminalInfo.alpha = make_float3(1.0f);
    terminalInfo.pathLength = 1;
    terminalInfo.hasQuery = materialSlot != 0xFFFFFFFF;
    terminalInfo.isTrainingPixel = false;
    terminalInfo.isUnbiasedTile = false;
    plp.s->inferenceTerminalInfoBuffer[linearIndex] = terminalInfo;
}
