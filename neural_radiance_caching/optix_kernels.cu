#include "hip/hip_runtime.h"
﻿#include "neural_radiance_caching_shared.h"

using namespace shared;

struct HitPointParameter {
    float b1, b2;
    int32_t primIndex;

    CUDA_DEVICE_FUNCTION static HitPointParameter get() {
        HitPointParameter ret;
        float2 bc = optixGetTriangleBarycentrics();
        ret.b1 = bc.x;
        ret.b2 = bc.y;
        ret.primIndex = optixGetPrimitiveIndex();
        return ret;
    }
};

struct HitGroupSBTRecordData {
    GeometryInstanceData geomInstData;

    CUDA_DEVICE_FUNCTION static const HitGroupSBTRecordData &get() {
        return *reinterpret_cast<HitGroupSBTRecordData*>(optixGetSbtDataPointer());
    }
};



CUDA_DEVICE_KERNEL void RT_AH_NAME(visibility)() {
    float visibility = 0.0f;
    VisibilityRayPayloadSignature::set(&visibility);
}



CUDA_DEVICE_KERNEL void RT_RG_NAME(setupGBuffers)() {
    uint2 launchIndex = make_uint2(optixGetLaunchIndex().x, optixGetLaunchIndex().y);

    const PerspectiveCamera &camera = plp.f->camera;
    float jx = 0.5f;
    float jy = 0.5f;
    if (plp.f->enableJittering) {
        // JP: ジッターをかけると現状の実装ではUnbiased要件を満たさないかもしれない。要検討。
        // EN: Jittering may break the requirements for unbiasedness with the current implementation.
        //     Need more consideration.
        PCG32RNG rng = plp.s->rngBuffer.read(launchIndex);
        jx = rng.getFloat0cTo1o();
        jy = rng.getFloat0cTo1o();
        plp.s->rngBuffer.write(launchIndex, rng);
    }
    float x = (launchIndex.x + jx) / plp.s->imageSize.x;
    float y = (launchIndex.y + jy) / plp.s->imageSize.y;
    float vh = 2 * std::tan(camera.fovY * 0.5f);
    float vw = camera.aspect * vh;

    float3 origin = camera.position;
    float3 direction = normalize(camera.orientation * make_float3(vw * (0.5f - x), vh * (0.5f - y), 1));

    HitPointParams hitPointParams;
    hitPointParams.positionInWorld = make_float3(NAN);
    hitPointParams.prevPositionInWorld = make_float3(NAN);
    hitPointParams.normalInWorld = make_float3(NAN);
    hitPointParams.texCoord = make_float2(NAN);
    hitPointParams.materialSlot = 0xFFFFFFFF;

    PickInfo pickInfo = {};

    HitPointParams* hitPointParamsPtr = &hitPointParams;
    PickInfo* pickInfoPtr = &pickInfo;
    optixu::trace<PrimaryRayPayloadSignature>(
        plp.f->travHandle, origin, direction,
        0.0f, FLT_MAX, 0.0f, 0xFF, OPTIX_RAY_FLAG_NONE,
        RayType_Primary, NumRayTypes, RayType_Primary,
        hitPointParamsPtr, pickInfoPtr);



    float2 curRasterPos = make_float2(launchIndex.x + 0.5f, launchIndex.y + 0.5f);
    float2 prevRasterPos =
        plp.f->prevCamera.calcScreenPosition(hitPointParams.prevPositionInWorld)
        * make_float2(plp.s->imageSize.x, plp.s->imageSize.y);
    float2 motionVector = curRasterPos - prevRasterPos;
    if (plp.f->resetFlowBuffer || isnan(hitPointParams.prevPositionInWorld.x))
        motionVector = make_float2(0.0f, 0.0f);

    GBuffer0 gBuffer0;
    gBuffer0.positionInWorld = hitPointParams.positionInWorld;
    gBuffer0.texCoord_x = hitPointParams.texCoord.x;
    GBuffer1 gBuffer1;
    gBuffer1.normalInWorld = hitPointParams.normalInWorld;
    gBuffer1.texCoord_y = hitPointParams.texCoord.y;
    GBuffer2 gBuffer2;
    gBuffer2.motionVector = motionVector;
    gBuffer2.materialSlot = hitPointParams.materialSlot;

    uint32_t bufIdx = plp.f->bufferIndex;
    plp.s->GBuffer0[bufIdx].write(launchIndex, gBuffer0);
    plp.s->GBuffer1[bufIdx].write(launchIndex, gBuffer1);
    plp.s->GBuffer2[bufIdx].write(launchIndex, gBuffer2);

    if (launchIndex.x == plp.f->mousePosition.x &&
        launchIndex.y == plp.f->mousePosition.y)
        *plp.f->pickInfo = pickInfo;

    // JP: デノイザーに必要な情報を出力。
    // EN: Output information required for the denoiser.
    float3 firstHitNormal = transpose(camera.orientation) * hitPointParams.normalInWorld;
    firstHitNormal.x *= -1;
    float3 prevAlbedoResult = make_float3(0.0f, 0.0f, 0.0f);
    float3 prevNormalResult = make_float3(0.0f, 0.0f, 0.0f);
    if (plp.f->numAccumFrames > 0) {
        prevAlbedoResult = getXYZ(plp.s->albedoAccumBuffer.read(launchIndex));
        prevNormalResult = getXYZ(plp.s->normalAccumBuffer.read(launchIndex));
    }
    float curWeight = 1.0f / (1 + plp.f->numAccumFrames);
    float3 albedoResult = (1 - curWeight) * prevAlbedoResult + curWeight * hitPointParams.albedo;
    float3 normalResult = (1 - curWeight) * prevNormalResult + curWeight * firstHitNormal;
    plp.s->albedoAccumBuffer.write(launchIndex, make_float4(albedoResult, 1.0f));
    plp.s->normalAccumBuffer.write(launchIndex, make_float4(normalResult, 1.0f));
}

CUDA_DEVICE_KERNEL void RT_CH_NAME(setupGBuffers)() {
    uint2 launchIndex = make_uint2(optixGetLaunchIndex().x, optixGetLaunchIndex().y);

    auto sbtr = HitGroupSBTRecordData::get();
    const InstanceData &inst = plp.f->instanceDataBuffer[optixGetInstanceId()];
    const GeometryInstanceData &geomInst = sbtr.geomInstData;

    HitPointParams* hitPointParams;
    PickInfo* pickInfo;
    PrimaryRayPayloadSignature::get(&hitPointParams, &pickInfo);

    auto hp = HitPointParameter::get();
    float3 positionInWorld;
    float3 prevPositionInWorld;
    float3 shadingNormalInWorld;
    float3 texCoord0DirInWorld;
    //float3 geometricNormalInWorld;
    float2 texCoord;
    {
        const Triangle &tri = geomInst.triangleBuffer[hp.primIndex];
        const Vertex &v0 = geomInst.vertexBuffer[tri.index0];
        const Vertex &v1 = geomInst.vertexBuffer[tri.index1];
        const Vertex &v2 = geomInst.vertexBuffer[tri.index2];
        float b1 = hp.b1;
        float b2 = hp.b2;
        float b0 = 1 - (b1 + b2);
        float3 localP = b0 * v0.position + b1 * v1.position + b2 * v2.position;
        shadingNormalInWorld = b0 * v0.normal + b1 * v1.normal + b2 * v2.normal;
        texCoord0DirInWorld = b0 * v0.texCoord0Dir + b1 * v1.texCoord0Dir + b2 * v2.texCoord0Dir;
        //geometricNormalInWorld = cross(v1.position - v0.position, v2.position - v0.position);
        texCoord = b0 * v0.texCoord + b1 * v1.texCoord + b2 * v2.texCoord;

        positionInWorld = optixTransformPointFromObjectToWorldSpace(localP);
        prevPositionInWorld = inst.prevTransform * localP;
        shadingNormalInWorld = normalize(optixTransformNormalFromObjectToWorldSpace(shadingNormalInWorld));
        texCoord0DirInWorld = normalize(optixTransformVectorFromObjectToWorldSpace(texCoord0DirInWorld));
        //geometricNormalInWorld = normalize(optixTransformNormalFromObjectToWorldSpace(geometricNormalInWorld));
        if (!allFinite(shadingNormalInWorld)) {
            shadingNormalInWorld = make_float3(0, 0, 1);
            texCoord0DirInWorld = make_float3(1, 0, 0);
        }
    }

    const MaterialData &mat = plp.s->materialDataBuffer[geomInst.materialSlot];

    BSDF bsdf;
    bsdf.setup(mat, texCoord);
    ReferenceFrame shadingFrame(shadingNormalInWorld, texCoord0DirInWorld);
    float3 modLocalNormal = mat.readModifiedNormal(mat.normal, texCoord, mat.normalDimension);
    if (plp.f->enableBumpMapping)
        applyBumpMapping(modLocalNormal, &shadingFrame);
    float3 vOut = -optixGetWorldRayDirection();
    float3 vOutLocal = shadingFrame.toLocal(normalize(vOut));

    hitPointParams->albedo = bsdf.evaluateDHReflectanceEstimate(vOutLocal);
    hitPointParams->positionInWorld = positionInWorld;
    hitPointParams->prevPositionInWorld = prevPositionInWorld;
    hitPointParams->normalInWorld = shadingFrame.normal;
    hitPointParams->texCoord = texCoord;
    hitPointParams->materialSlot = geomInst.materialSlot;

    // JP: マウスが乗っているピクセルの情報を出力する。
    // EN: Export the information of the pixel on which the mouse is.
    if (launchIndex.x == plp.f->mousePosition.x &&
        launchIndex.y == plp.f->mousePosition.y) {
        pickInfo->hit = true;
        pickInfo->instSlot = optixGetInstanceId();
        pickInfo->geomInstSlot = geomInst.geomInstSlot;
        pickInfo->matSlot = geomInst.materialSlot;
        pickInfo->primIndex = hp.primIndex;
        pickInfo->positionInWorld = positionInWorld;
        pickInfo->normalInWorld = shadingFrame.normal;
        pickInfo->albedo = hitPointParams->albedo;
        float3 emittance = make_float3(0.0f, 0.0f, 0.0f);
        if (mat.emittance) {
            float4 texValue = tex2DLod<float4>(mat.emittance, texCoord.x, texCoord.y, 0.0f);
            emittance = make_float3(texValue);
        }
        pickInfo->emittance = emittance;
    }
}

CUDA_DEVICE_KERNEL void RT_MS_NAME(setupGBuffers)() {
    uint2 launchIndex = make_uint2(optixGetLaunchIndex().x, optixGetLaunchIndex().y);

    float3 vOut = -optixGetWorldRayDirection();
    float3 p = -vOut;

    float posPhi, posTheta;
    toPolarYUp(p, &posPhi, &posTheta);

    float phi = posPhi + plp.f->envLightRotation;

    float u = phi / (2 * Pi);
    u -= floorf(u);
    float v = posTheta / Pi;

    HitPointParams* hitPointParams;
    PickInfo* pickInfo;
    PrimaryRayPayloadSignature::get(&hitPointParams, &pickInfo);

    hitPointParams->albedo = make_float3(0.0f, 0.0f, 0.0f);
    hitPointParams->positionInWorld = p;
    hitPointParams->prevPositionInWorld = p;
    hitPointParams->normalInWorld = vOut;
    hitPointParams->texCoord = make_float2(u, v);
    hitPointParams->materialSlot = 0xFFFFFFFF;

    // JP: マウスが乗っているピクセルの情報を出力する。
    // EN: Export the information of the pixel on which the mouse is.
    if (launchIndex.x == plp.f->mousePosition.x &&
        launchIndex.y == plp.f->mousePosition.y) {
        pickInfo->hit = true;
        pickInfo->instSlot = 0xFFFFFFFF;
        pickInfo->geomInstSlot = 0xFFFFFFFF;
        pickInfo->matSlot = 0xFFFFFFFF;
        pickInfo->primIndex = 0xFFFFFFFF;
        pickInfo->positionInWorld = p;
        pickInfo->albedo = make_float3(0.0f, 0.0f, 0.0f);
        float3 emittance = make_float3(0.0f, 0.0f, 0.0f);
        if (plp.s->envLightTexture && plp.f->enableEnvLight) {
            float4 texValue = tex2DLod<float4>(plp.s->envLightTexture, u, v, 0.0f);
            emittance = make_float3(texValue);
            emittance *= Pi * plp.f->envLightPowerCoeff;
        }
        pickInfo->emittance = emittance;
        pickInfo->normalInWorld = vOut;
    }
}



CUDA_DEVICE_FUNCTION void convertToPolar(const float3 &dir, float* phi, float* theta) {
    float z = std::fmin(std::fmax(dir.z, -1.0f), 1.0f);
    *theta = std::acos(z);
    *phi = std::atan2(dir.y, dir.x);
}

CUDA_DEVICE_FUNCTION void createRadianceQuery(
    const float3 &positionInWorld, const float3 &normalInWorld, const float3 &scatteredDirInWorld,
    float roughness, const float3 &diffuseReflectance, const float3 &specularReflectance,
    RadianceQuery* query) {
    float phi, theta;
    query->position = positionInWorld;
    convertToPolar(normalInWorld, &phi, &theta);
    query->normal_phi = phi;
    query->normal_theta = theta;
    convertToPolar(scatteredDirInWorld, &phi, &theta);
    query->vOut_phi = phi;
    query->vOut_theta = theta;
    query->roughness = roughness;
    query->diffuseReflectance = diffuseReflectance;
    query->specularReflectance = specularReflectance;
}

static constexpr bool useSolidAngleSampling = false;
static constexpr bool useImplicitLightSampling = true;
static constexpr bool useExplicitLightSampling = true;
static constexpr bool useMultipleImportanceSampling = useImplicitLightSampling && useExplicitLightSampling;
static_assert(useImplicitLightSampling || useExplicitLightSampling, "Invalid configuration for light sampling.");

CUDA_DEVICE_FUNCTION float3 performNextEventEstimation(
    const float3 &shadingPoint, const float3 &vOutLocal, const ReferenceFrame &shadingFrame, const BSDF &bsdf,
    PCG32RNG &rng) {
    float3 ret = make_float3(0.0f);
    if constexpr (useExplicitLightSampling) {
        float uLight = rng.getFloat0cTo1o();
        bool selectEnvLight = false;
        float probToSampleCurLightType = 1.0f;
        if (plp.s->envLightTexture && plp.f->enableEnvLight) {
            if (uLight < probToSampleEnvLight) {
                probToSampleCurLightType = probToSampleEnvLight;
                uLight /= probToSampleCurLightType;
                selectEnvLight = true;
            }
            else {
                probToSampleCurLightType = 1.0f - probToSampleEnvLight;
                uLight = (uLight - probToSampleEnvLight) / probToSampleCurLightType;
            }
        }
        LightSample lightSample;
        float areaPDensity;
        sampleLight<useSolidAngleSampling>(
            shadingPoint,
            uLight, selectEnvLight, rng.getFloat0cTo1o(), rng.getFloat0cTo1o(),
            &lightSample, &areaPDensity);
        areaPDensity *= probToSampleCurLightType;
        float misWeight = 1.0f;
        if constexpr (useMultipleImportanceSampling) {
            float3 shadowRay = lightSample.atInfinity ?
                lightSample.position :
                (lightSample.position - shadingPoint);
            float dist2 = sqLength(shadowRay);
            shadowRay /= std::sqrt(dist2);
            float3 vInLocal = shadingFrame.toLocal(shadowRay);
            float lpCos = std::fabs(dot(shadowRay, lightSample.normal));
            float bsdfPDensity = bsdf.evaluatePDF(vOutLocal, vInLocal) * lpCos / dist2;
            if (!isfinite(bsdfPDensity))
                bsdfPDensity = 0.0f;
            float lightPDensity = areaPDensity;
            misWeight = pow2(lightPDensity) / (pow2(bsdfPDensity) + pow2(lightPDensity));
        }
        if (areaPDensity > 0.0f)
            ret = performDirectLighting<true>(
                shadingPoint, vOutLocal, shadingFrame, bsdf, lightSample) * (misWeight / areaPDensity);
    }

    return ret;
}

CUDA_DEVICE_FUNCTION void pathTrace_rayGen_generic() {
    uint2 launchIndex = make_uint2(optixGetLaunchIndex().x, optixGetLaunchIndex().y);

    uint32_t bufIdx = plp.f->bufferIndex;
    GBuffer0 gBuffer0 = plp.s->GBuffer0[bufIdx].read(launchIndex);
    GBuffer1 gBuffer1 = plp.s->GBuffer1[bufIdx].read(launchIndex);
    GBuffer2 gBuffer2 = plp.s->GBuffer2[bufIdx].read(launchIndex);

    float3 positionInWorld = gBuffer0.positionInWorld;
    float3 shadingNormalInWorld = gBuffer1.normalInWorld;
    float2 texCoord = make_float2(gBuffer0.texCoord_x, gBuffer1.texCoord_y);
    uint32_t materialSlot = gBuffer2.materialSlot;

    const PerspectiveCamera &camera = plp.f->camera;

    uint32_t linearTileIndex;
    const uint2 tileSize = *plp.s->tileSize;
    const uint32_t numPixelsInTile = tileSize.x * tileSize.y;
    {
        uint2 numTiles = (plp.s->imageSize + tileSize - 1) / tileSize;
        uint2 tileIndex = launchIndex / tileSize;
        linearTileIndex = tileIndex.y * numTiles.x + tileIndex.x;
    }

    bool useEnvLight = plp.s->envLightTexture && plp.f->enableEnvLight;
    float3 contribution = make_float3(0.001f, 0.001f, 0.001f);
    bool renderingPathEndsWithCache = false;
    uint32_t pathLength = 1;
    if (materialSlot != 0xFFFFFFFF) {
        bool isTrainingPath;
        {
            uint2 localIndex = launchIndex % tileSize;
            uint32_t localLinearIndex = localIndex.y * tileSize.x + localIndex.x;
            isTrainingPath = (localLinearIndex + *plp.s->offsetToSelectTrainingPath) % numPixelsInTile == 0;
        }

        float3 alpha = make_float3(1.0f);
        float initImportance = sRGB_calcLuminance(alpha);
        PCG32RNG rng = plp.s->rngBuffer.read(launchIndex);

        // JP: 最初の交点におけるシェーディング。
        // EN: Shading on the first hit.
        float3 vIn;
        float dirPDensity;
        float primaryPathSpread;
        float3 localThroughput;
        uint32_t trainDataIndex;
        {
            const MaterialData &mat = plp.s->materialDataBuffer[materialSlot];

            // TODO?: Use true geometric normal.
            float3 geometricNormalInWorld = shadingNormalInWorld;
            float3 vOut = camera.position - positionInWorld;
            float primaryDist2 = sqLength(vOut);
            vOut /= std::sqrt(primaryDist2);
            float primaryDotVN = dot(vOut, geometricNormalInWorld);
            float frontHit = primaryDotVN >= 0.0f ? 1.0f : -1.0f;

            primaryPathSpread = primaryDist2 / (4 * Pi * std::fabs(primaryDotVN));

            ReferenceFrame shadingFrame(shadingNormalInWorld);
            positionInWorld = offsetRayOriginNaive(positionInWorld, frontHit * geometricNormalInWorld);
            float3 vOutLocal = shadingFrame.toLocal(vOut);

            // JP: 光源を直接見ている場合の寄与を蓄積。
            // EN: Accumulate the contribution from a light source directly seeing.
            contribution = make_float3(0.0f);
            if (vOutLocal.z > 0 && mat.emittance) {
                float4 texValue = tex2DLod<float4>(mat.emittance, texCoord.x, texCoord.y, 0.0f);
                float3 emittance = make_float3(texValue);
                contribution += alpha * emittance / Pi;
            }

            BSDF bsdf;
            bsdf.setup(mat, texCoord);

            // Next event estimation (explicit light sampling) on the first hit.
            float3 directContNEE = performNextEventEstimation(
                positionInWorld, vOutLocal, shadingFrame, bsdf, rng);
            contribution += alpha * directContNEE;

            // generate a next ray.
            float3 vInLocal;
            localThroughput = bsdf.sampleThroughput(
                vOutLocal, rng.getFloat0cTo1o(), rng.getFloat0cTo1o(),
                &vInLocal, &dirPDensity);
            alpha *= localThroughput;
            vIn = shadingFrame.fromLocal(vInLocal);

            // JP: 訓練データエントリーの確保。
            // EN: 
            if (isTrainingPath) {
                trainDataIndex = atomicAdd(plp.s->numTrainingData, 1u);

                if (trainDataIndex < maxNumTrainingDataPerFrame) {
                    float roughness;
                    float3 diffuseReflectance, specularReflectance;
                    bsdf.getSurfaceParameters(
                        &diffuseReflectance, &specularReflectance, &roughness);

                    RadianceQuery radQuery;
                    createRadianceQuery(
                        positionInWorld, shadingFrame.normal, vOut,
                        roughness, diffuseReflectance, specularReflectance,
                        &radQuery);
                    plp.s->trainRadianceQueryBuffer[trainDataIndex] = radQuery;

                    TrainingVertexInfo vertInfo;
                    vertInfo.localThroughput = localThroughput;
                    vertInfo.prevVertexDataIndex = invalidVertexDataIndex;
                    plp.s->trainVertexInfoBuffer[trainDataIndex] = vertInfo;

                    // JP: 現在の頂点に対する直接照明(NEE)によるScattered Radianceでターゲット値を初期化。
                    // EN:
                    plp.s->trainTargetBuffer[trainDataIndex] = directContNEE;
                }
                else {
                    trainDataIndex = invalidVertexDataIndex;
                }
            }
        }

        // Path extension loop
        PathTraceWriteOnlyPayload woPayload = {};
        PathTraceWriteOnlyPayload* woPayloadPtr = &woPayload;
        PathTraceReadWritePayload rwPayload = {};
        PathTraceReadWritePayload* rwPayloadPtr = &rwPayload;
        rwPayload.rng = rng;
        rwPayload.initImportance = initImportance;
        rwPayload.alpha = alpha;
        rwPayload.contribution = contribution;
        rwPayload.prevDirPDensity = dirPDensity;
        rwPayload.linearTileIndex = linearTileIndex;
        rwPayload.primaryPathSpread = primaryPathSpread;
        rwPayload.curSqrtPathSpread = 0.0f;
        rwPayload.prevLocalThroughput = localThroughput;
        rwPayload.prevTrainDataIndex = trainDataIndex;
        rwPayload.renderingPathEndsWithCache = false;
        rwPayload.isTrainingPath = isTrainingPath;
        rwPayload.trainingSuffixEndsWithCache = false;
        rwPayload.pathLength = 1;
        float3 rayOrg = positionInWorld;
        float3 rayDir = vIn;
        while (true) {
            bool isValidSampling = rwPayload.prevDirPDensity > 0.0f && isfinite(rwPayload.prevDirPDensity);
            if (!isValidSampling)
                break;

            ++rwPayload.pathLength;
            if (rwPayload.pathLength >= plp.f->maxPathLength)
                rwPayload.maxLengthTerminate = true;
            rwPayload.terminate = true;
            // JP: 経路長制限に到達したときに、implicit light samplingを使わない場合はClosest-hit program内
            //     で行うことが無いので終了する。
            // EN: Nothing to do in the closest-hit program when reaching the path length limit
            //     in the case implicit light sampling is unused.
            if constexpr (!useImplicitLightSampling) {
                if (rwPayload.maxLengthTerminate)
                    break;
                // Russian roulette
                float continueProb = std::fmin(sRGB_calcLuminance(rwPayload.alpha) / rwPayload.initImportance, 1.0f);
                if (rwPayload.rng.getFloat0cTo1o() >= continueProb)
                    break;
                rwPayload.alpha /= continueProb;
            }

            constexpr RayType pathTraceRayType = RayType_PathTrace;
            optixu::trace<PathTraceRayPayloadSignature>(
                plp.f->travHandle, rayOrg, rayDir,
                0.0f, FLT_MAX, 0.0f, 0xFF, OPTIX_RAY_FLAG_NONE,
                pathTraceRayType, NumRayTypes, pathTraceRayType,
                woPayloadPtr, rwPayloadPtr);
            if (rwPayload.terminate)
                break;
            rayOrg = woPayload.nextOrigin;
            rayDir = woPayload.nextDirection;
        }
        contribution = rwPayload.contribution;

        plp.s->rngBuffer.write(launchIndex, rwPayload.rng);

        renderingPathEndsWithCache = rwPayload.renderingPathEndsWithCache;
        pathLength = rwPayload.pathLength;
        if (rwPayload.isTrainingPath && !rwPayload.trainingSuffixEndsWithCache) {
            TrainingSuffixTerminalInfo terminalInfo;
            terminalInfo.prevVertexDataIndex = rwPayload.prevTrainDataIndex;
            terminalInfo.hasQuery = false;
            plp.s->trainSuffixTerminalInfoBuffer[rwPayload.linearTileIndex] = terminalInfo;
        }
    }
    else {
        // JP: 環境光源を直接見ている場合の寄与を蓄積。
        // EN: Accumulate the contribution from the environmental light source directly seeing.
        if (useEnvLight) {
            float u = texCoord.x, v = texCoord.y;
            float4 texValue = tex2DLod<float4>(plp.s->envLightTexture, u, v, 0.0f);
            float3 luminance = plp.f->envLightPowerCoeff * make_float3(texValue);
            contribution = luminance;
        }
    }

    uint32_t linearIndex = launchIndex.y * plp.s->imageSize.x + launchIndex.x;

    //// JP: 
    //// EN: 
    //RadianceQuery radQuery;
    //radQuery.position = make_float3(0.0f, 0.0f, 0.0f);
    //radQuery.vOut = make_float3(0.0f, 0.0f, 0.0f);
    //radQuery.normal = make_float3(0.0f, 0.0f, 0.0f);
    //radQuery.diffuseReflectance = make_float3(0.0f, 0.0f, 0.0f);
    //radQuery.specularReflectance = make_float3(0.0f, 0.0f, 0.0f);
    //radQuery.roughness = 0.0f;
    //plp.s->inferenceRadianceQueryBuffer[linearIndex] = radQuery;

    // JP: 無限遠にレイが飛んだか、ロシアンルーレットによってパストレースが完了したケース。
    // EN: 
    if (!renderingPathEndsWithCache) {
        TerminalInfo terminalInfo;
        terminalInfo.alpha = make_float3(0.0f, 0.0f, 0.0f);
        terminalInfo.pathLength = pathLength;
        terminalInfo.hasQuery = false;
        plp.s->inferenceTerminalInfoBuffer[linearIndex] = terminalInfo;
    }

    plp.s->perFrameContributionBuffer[linearIndex] = contribution;
}

CUDA_DEVICE_FUNCTION void pathTrace_closestHit_generic() {
    uint2 launchIndex = make_uint2(optixGetLaunchIndex().x, optixGetLaunchIndex().y);

    auto sbtr = HitGroupSBTRecordData::get();
    const InstanceData &inst = plp.f->instanceDataBuffer[optixGetInstanceId()];
    const GeometryInstanceData &geomInst = sbtr.geomInstData;

    PathTraceWriteOnlyPayload* woPayload;
    PathTraceReadWritePayload* rwPayload;
    PathTraceRayPayloadSignature::get(&woPayload, &rwPayload);
    PCG32RNG &rng = rwPayload->rng;

    const float3 rayOrigin = optixGetWorldRayOrigin();

    auto hp = HitPointParameter::get();
    float3 positionInWorld;
    float3 shadingNormalInWorld;
    float3 texCoord0DirInWorld;
    float3 geometricNormalInWorld;
    float2 texCoord;
    float hypAreaPDensity;
    computeSurfacePoint<useMultipleImportanceSampling, useSolidAngleSampling>(
        inst, geomInst, hp.primIndex, hp.b1, hp.b2,
        rayOrigin,
        &positionInWorld, &shadingNormalInWorld, &texCoord0DirInWorld,
        &geometricNormalInWorld, &texCoord, &hypAreaPDensity);
    if constexpr (!useMultipleImportanceSampling)
        (void)hypAreaPDensity;

    const MaterialData &mat = plp.s->materialDataBuffer[geomInst.materialSlot];

    float3 vOut = normalize(-optixGetWorldRayDirection());
    float frontHit = dot(vOut, geometricNormalInWorld) >= 0.0f ? 1.0f : -1.0f;

    ReferenceFrame shadingFrame(shadingNormalInWorld, texCoord0DirInWorld);
    float3 modLocalNormal = mat.readModifiedNormal(mat.normal, texCoord, mat.normalDimension);
    if (plp.f->enableBumpMapping)
        applyBumpMapping(modLocalNormal, &shadingFrame);
    positionInWorld = offsetRayOrigin(positionInWorld, frontHit * geometricNormalInWorld);
    float3 vOutLocal = shadingFrame.toLocal(vOut);

    float dist2 = squaredDistance(rayOrigin, positionInWorld);
    rwPayload->curSqrtPathSpread += std::sqrt(dist2 / (rwPayload->prevDirPDensity * std::fabs(vOutLocal.z)));

    if constexpr (useImplicitLightSampling) {
        // Implicit Light Sampling
        if (vOutLocal.z > 0 && mat.emittance) {
            float4 texValue = tex2DLod<float4>(mat.emittance, texCoord.x, texCoord.y, 0.0f);
            float3 emittance = make_float3(texValue);
            float misWeight = 1.0f;
            if constexpr (useMultipleImportanceSampling) {
                float lightPDensity = hypAreaPDensity * dist2 / vOutLocal.z;
                float bsdfPDensity = rwPayload->prevDirPDensity;
                misWeight = pow2(bsdfPDensity) / (pow2(bsdfPDensity) + pow2(lightPDensity));
            }
            float3 directContImplicit = emittance * (misWeight / Pi);
            rwPayload->contribution += rwPayload->alpha * directContImplicit;

            // JP: 1つ前の頂点に対する直接照明(Implicit)によるScattered Radianceをターゲット値に加算。
            // EN: 
            if (rwPayload->isTrainingPath && rwPayload->prevTrainDataIndex != invalidVertexDataIndex) {
                plp.s->trainTargetBuffer[rwPayload->prevTrainDataIndex] +=
                    rwPayload->prevLocalThroughput * directContImplicit;
            }
        }

        // Russian roulette
        float continueProb = std::fmin(sRGB_calcLuminance(rwPayload->alpha) / rwPayload->initImportance, 1.0f);
        if (rng.getFloat0cTo1o() >= continueProb || rwPayload->maxLengthTerminate)
            return;
        rwPayload->alpha /= continueProb;
    }

    BSDF bsdf;
    bsdf.setup(mat, texCoord);

    // Path termination into the neural radiance cache.
    bool pathIsSpreadEnough =
        pow2(rwPayload->curSqrtPathSpread) > pathTerminationFactor * rwPayload->primaryPathSpread;
    if (pathIsSpreadEnough) {
        uint32_t linearIndex = launchIndex.y * plp.s->imageSize.x + launchIndex.x;

        float roughness;
        float3 diffuseReflectance, specularReflectance;
        bsdf.getSurfaceParameters(
            &diffuseReflectance, &specularReflectance, &roughness);

        // JP: Radianceクエリーのための情報を記録する。
        // EN: 
        RadianceQuery radQuery;
        createRadianceQuery(
            positionInWorld, shadingFrame.normal, vOut,
            roughness, diffuseReflectance, specularReflectance,
            &radQuery);

        if (!rwPayload->renderingPathEndsWithCache) {
            plp.s->inferenceRadianceQueryBuffer[linearIndex] = radQuery;

            TerminalInfo terminalInfo;
            terminalInfo.alpha = rwPayload->alpha;
            terminalInfo.pathLength = rwPayload->pathLength;
            terminalInfo.hasQuery = true;
            plp.s->inferenceTerminalInfoBuffer[linearIndex] = terminalInfo;

            rwPayload->renderingPathEndsWithCache = true;
            if (rwPayload->isTrainingPath)
                rwPayload->curSqrtPathSpread = 0;
            else
                return;
        }
        else {
            // JP: 訓練データを既に十分生成している場合は既に強制的にTraining Suffixは終了したことになっている。
            if (!rwPayload->trainingSuffixEndsWithCache) {
                // JP: 
                // EN: 
                uint32_t offset = plp.s->imageSize.x * plp.s->imageSize.y;
                plp.s->inferenceRadianceQueryBuffer[offset + rwPayload->linearTileIndex] = radQuery;

                TrainingSuffixTerminalInfo terminalInfo;
                terminalInfo.prevVertexDataIndex = rwPayload->prevTrainDataIndex;
                terminalInfo.hasQuery = true;
                plp.s->trainSuffixTerminalInfoBuffer[rwPayload->linearTileIndex] = terminalInfo;

                rwPayload->trainingSuffixEndsWithCache = true;
            }
            return;
        }
    }

    // Next Event Estimation (Explicit Light Sampling)
    float3 directContNEE = performNextEventEstimation(
        positionInWorld, vOutLocal, shadingFrame, bsdf, rng);
    rwPayload->contribution += rwPayload->alpha * directContNEE;

    // generate a next ray.
    float3 vInLocal;
    float dirPDensity;
    float3 localThroughput = bsdf.sampleThroughput(
        vOutLocal, rng.getFloat0cTo1o(), rng.getFloat0cTo1o(),
        &vInLocal, &dirPDensity);
    rwPayload->alpha *= localThroughput;
    float3 vIn = shadingFrame.fromLocal(vInLocal);

    woPayload->nextOrigin = positionInWorld;
    woPayload->nextDirection = vIn;
    rwPayload->prevDirPDensity = dirPDensity;
    rwPayload->prevLocalThroughput = localThroughput;
    rwPayload->terminate = false;

    // JP: 訓練データエントリーの確保。
    // EN: 
    if (rwPayload->isTrainingPath && !rwPayload->trainingSuffixEndsWithCache) {
        uint32_t trainDataIndex = atomicAdd(plp.s->numTrainingData, 1u);

        float roughness;
        float3 diffuseReflectance, specularReflectance;
        bsdf.getSurfaceParameters(
            &diffuseReflectance, &specularReflectance, &roughness);

        RadianceQuery radQuery;
        createRadianceQuery(
            positionInWorld, shadingFrame.normal, vOut,
            roughness, diffuseReflectance, specularReflectance,
            &radQuery);

        if (trainDataIndex < maxNumTrainingDataPerFrame) {
            plp.s->trainRadianceQueryBuffer[trainDataIndex] = radQuery;

            TrainingVertexInfo vertInfo;
            vertInfo.localThroughput = localThroughput;
            vertInfo.prevVertexDataIndex = rwPayload->prevTrainDataIndex;
            plp.s->trainVertexInfoBuffer[trainDataIndex] = vertInfo;

            // JP: 現在の頂点に対する直接照明(NEE)によるScattered Radianceでターゲット値を初期化。
            // EN:
            plp.s->trainTargetBuffer[trainDataIndex] = directContNEE;

            rwPayload->prevTrainDataIndex = trainDataIndex;
        }
        else {
            // JP: 
            // EN: 
            uint32_t offset = plp.s->imageSize.x * plp.s->imageSize.y;
            plp.s->inferenceRadianceQueryBuffer[offset + rwPayload->linearTileIndex] = radQuery;

            TrainingSuffixTerminalInfo terminalInfo;
            terminalInfo.prevVertexDataIndex = rwPayload->prevTrainDataIndex;
            terminalInfo.hasQuery = true;
            plp.s->trainSuffixTerminalInfoBuffer[rwPayload->linearTileIndex] = terminalInfo;

            rwPayload->trainingSuffixEndsWithCache = true;
        }
    }
}

CUDA_DEVICE_KERNEL void RT_RG_NAME(pathTrace)() {
    pathTrace_rayGen_generic();
}

CUDA_DEVICE_KERNEL void RT_CH_NAME(pathTrace)() {
    pathTrace_closestHit_generic();
}

CUDA_DEVICE_KERNEL void RT_MS_NAME(pathTrace)() {
    if constexpr (useImplicitLightSampling) {
        if (!plp.s->envLightTexture || !plp.f->enableEnvLight)
            return;

        PathTraceReadWritePayload* rwPayload;
        PathTraceRayPayloadSignature::get(nullptr, &rwPayload);

        float3 rayDir = normalize(optixGetWorldRayDirection());
        float posPhi, theta;
        toPolarYUp(rayDir, &posPhi, &theta);

        float phi = posPhi + plp.f->envLightRotation;
        phi = phi - floorf(phi / (2 * Pi)) * 2 * Pi;
        float2 texCoord = make_float2(phi / (2 * Pi), theta / Pi);

        // Implicit Light Sampling
        float4 texValue = tex2DLod<float4>(plp.s->envLightTexture, texCoord.x, texCoord.y, 0.0f);
        float3 luminance = plp.f->envLightPowerCoeff * make_float3(texValue);
        float misWeight = 1.0f;
        if constexpr (useMultipleImportanceSampling) {
            float uvPDF = plp.s->envLightImportanceMap.evaluatePDF(texCoord.x, texCoord.y);
            float hypAreaPDensity = uvPDF / (2 * Pi * Pi * std::sin(theta));
            float lightPDensity = probToSampleEnvLight * hypAreaPDensity;
            float bsdfPDensity = rwPayload->prevDirPDensity;
            misWeight = pow2(bsdfPDensity) / (pow2(bsdfPDensity) + pow2(lightPDensity));
        }
        float3 directContImplicit = misWeight * luminance;
        rwPayload->contribution += rwPayload->alpha * directContImplicit;

        // JP: 1つ前の頂点に対する直接照明(Implicit)によるScattered Radianceをターゲット値に加算。
        // EN: 
        if (rwPayload->isTrainingPath) {
            plp.s->trainTargetBuffer[rwPayload->prevTrainDataIndex] +=
                rwPayload->prevLocalThroughput * directContImplicit;
        }
    }
}
