#include "network_interface.h"

#include <hip/hip_runtime.h>

#define TCNN_MIN_GPU_ARCH 86
#include <tiny-cuda-nn/common_device.h>
#include <tiny-cuda-nn/config.h>
#include <memory>

using namespace tcnn;
using precision_t = network_precision_t;

// Position: 3
// Scattered Direction: 2
// Normal: 2
// Roughness: 1
// Diffuse Reflectance: 3
// Specular Reflectance: 3
constexpr static uint32_t numInputDims = 14;
// RGB Radiance: 3
constexpr static uint32_t numOutputDims = 3;

class NeuralRadianceCache::Priv {
    std::shared_ptr<Loss<precision_t>> loss;
    std::shared_ptr<Optimizer<precision_t>> optimizer;
    std::shared_ptr<NetworkWithInputEncoding<precision_t>> network;

    std::shared_ptr<Trainer<float, precision_t, precision_t>> trainer;

public:
    friend class NeuralRadianceCache;

    Priv() {}
};



NeuralRadianceCache::NeuralRadianceCache() {
    m = new Priv();
}

NeuralRadianceCache::~NeuralRadianceCache() {
    delete m;
}

void NeuralRadianceCache::initialize() {
    json config = {
        {"loss", {
            {"otype", "RelativeL2Luminance"}
        }},
        {"optimizer", {
            {"otype", "EMA"},
            {"decay", 0.99f},
            {"nesteed", {
                {"otype", "Adam"},
                {"learning_rate", 1e-2f},
                {"beta1", 0.9f},
                {"beta2", 0.99f},
                {"l2_reg", 0.0f}
            }}
        }},
        {"encoding", {
            {"otype", "NRC"}
        }},
        {"network", {
            {"otype", "FullyFusedMLP"},
            {"n_neurons", 64},
            {"n_hidden_layers", 5},
            {"activation", "ReLU"},
            {"output_activation", "None"},
        }}
    };

    m->loss.reset(create_loss<precision_t>(config.value("loss", json::object())));
    m->optimizer.reset(create_optimizer<precision_t>(config.value("optimizer", json::object())));
    m->network = std::make_shared<NetworkWithInputEncoding<precision_t>>(
        numInputDims, numOutputDims,
        config.value("encoding", json::object()),
        config.value("network", json::object()));

    m->trainer = std::make_shared<Trainer<float, precision_t, precision_t>>(
        m->network, m->optimizer, m->loss);
}

void NeuralRadianceCache::finalize() {
    m->trainer = nullptr;
    m->network = nullptr;
    m->optimizer = nullptr;
    m->loss = nullptr;
}

void NeuralRadianceCache::infer(
    hipStream_t stream, float* inputData, uint32_t numData, float* predictionData) {
    uint32_t numDataPadded = (numData + 255) / 256 * 256;
    GPUMatrix<float> inputs(inputData, numInputDims, numDataPadded);
    GPUMatrix<float> predictions(predictionData, numOutputDims, numDataPadded);
    m->network->inference(stream, inputs, predictions);
}

void NeuralRadianceCache::train(
    hipStream_t stream, float* inputData, float* targetData, uint32_t numData) {
    uint32_t numDataPadded = (numData + 255) / 256 * 256;
    GPUMatrix<float> inputs(inputData, numInputDims, numDataPadded);
    GPUMatrix<float> targets(targetData, numOutputDims, numDataPadded);
    m->trainer->training_step(stream, inputs, targets, nullptr);
}
