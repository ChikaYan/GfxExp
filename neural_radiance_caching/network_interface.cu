#include "network_interface.h"
#include "../common/common_shared.h"

#include <hip/hip_runtime.h>

#define TCNN_MIN_GPU_ARCH 86
#include <tiny-cuda-nn/common_device.h>
#include <tiny-cuda-nn/config.h>
#include <memory>

using namespace tcnn;
using precision_t = network_precision_t;

// Position: 3
// Scattered Direction: 2
// Normal: 2
// Roughness: 1
// Diffuse Reflectance: 3
// Specular Reflectance: 3
constexpr static uint32_t numInputDims = 14;
// RGB Radiance: 3
constexpr static uint32_t numOutputDims = 3;

class NeuralRadianceCache::Priv {
    std::shared_ptr<Loss<precision_t>> loss;
    std::shared_ptr<Optimizer<precision_t>> optimizer;
    std::shared_ptr<NetworkWithInputEncoding<precision_t>> network;

    std::shared_ptr<Trainer<float, precision_t, precision_t>> trainer;

public:
    friend class NeuralRadianceCache;

    Priv() {}
};



NeuralRadianceCache::NeuralRadianceCache() {
    m = new Priv();
}

NeuralRadianceCache::~NeuralRadianceCache() {
    delete m;
}

void NeuralRadianceCache::initialize(PositionEncoding posEnc, uint32_t numHiddenLayers, float learningRate) {
    json config = {
        {"loss", {
            {"otype", "RelativeL2Luminance"}
        }},
        {"optimizer", {
            {"otype", "EMA"},
            {"decay", 0.99f},
            {"nested", {
                {"otype", "Adam"},
                {"learning_rate", learningRate},
            }}
        }},
        {"network", {
            {"otype", "FullyFusedMLP"},
            {"n_neurons", 64},
            {"n_hidden_layers", numHiddenLayers},
            {"activation", "ReLU"},
            {"output_activation", "None"},
        }}
    };

    if (posEnc == PositionEncoding::OneBlob) {
        config["encoding"] = { {"otype", "NRC"} };
    }
    else if (posEnc == PositionEncoding::Hash) {
        config["encoding"] = {
            {"otype", "Composite"},
            {"nested", {
                {
                    {"n_dims_to_encode", 3},
                    {"otype", "HashGrid"},
                    {"n_levels", 16},
                    {"n_features_per_level", 2},
                    {"log2_hashmap_size", 19},
                    {"base_resolution", 16},
                    {"per_level_scale", 1.5},
                },
                {
                    {"n_dims_to_encode", 5},
                    {"otype", "OneBlob"},
                    {"n_bins", 4},
                },
                {
                    {"otype", "Identity"}
                },
            }}
        };
        config["optimizer"]["nested"]["epsilon"] = 1e-15f;
    }

    m->loss.reset(create_loss<precision_t>(config.value("loss", json::object())));
    m->optimizer.reset(create_optimizer<precision_t>(config.value("optimizer", json::object())));
    m->network = std::make_shared<NetworkWithInputEncoding<precision_t>>(
        numInputDims, numOutputDims,
        config.value("encoding", json::object()),
        config.value("network", json::object()));

    m->trainer = std::make_shared<Trainer<float, precision_t, precision_t>>(
        m->network, m->optimizer, m->loss);
}

void NeuralRadianceCache::finalize() {
    m->trainer = nullptr;
    m->network = nullptr;
    m->optimizer = nullptr;
    m->loss = nullptr;
}

void NeuralRadianceCache::infer(
    hipStream_t stream, float* inputData, uint32_t numData, float* predictionData) {
    Assert((numData & 0x7F) == 0, "numData must be a multiple of 128.");
    GPUMatrix<float> inputs(inputData, numInputDims, numData);
    GPUMatrix<float> predictions(predictionData, numOutputDims, numData);
    m->network->inference(stream, inputs, predictions);
}

void NeuralRadianceCache::train(
    hipStream_t stream, float* inputData, float* targetData, uint32_t numData) {
    Assert((numData & 0x7F) == 0, "numData must be a multiple of 128.");
    GPUMatrix<float> inputs(inputData, numInputDims, numData);
    GPUMatrix<float> targets(targetData, numOutputDims, numData);
    m->trainer->training_step(stream, inputs, targets, nullptr);
}
