#include "network_interface.h"
#include "../common/common_shared.h"

#include <hip/hip_runtime.h>

#define TCNN_MIN_GPU_ARCH 86
#include <tiny-cuda-nn/common_device.h>
#include <tiny-cuda-nn/config.h>
#include <memory>

using namespace tcnn;
using precision_t = network_precision_t;

// Position: 3
// Scattered Direction: 2
// Normal: 2
// Roughness: 1
// Diffuse Reflectance: 3
// Specular Reflectance: 3
constexpr static uint32_t numInputDims = 14;
// RGB Radiance: 3
constexpr static uint32_t numOutputDims = 3;

class NeuralRadianceCache::Priv {
    std::shared_ptr<Loss<precision_t>> loss;
    std::shared_ptr<Optimizer<precision_t>> optimizer;
    std::shared_ptr<NetworkWithInputEncoding<precision_t>> network;

    std::shared_ptr<Trainer<float, precision_t, precision_t>> trainer;

public:
    friend class NeuralRadianceCache;

    Priv() {}
};



NeuralRadianceCache::NeuralRadianceCache() {
    m = new Priv();
}

NeuralRadianceCache::~NeuralRadianceCache() {
    delete m;
}

void NeuralRadianceCache::initialize(uint32_t numHiddenLayers, float learningRate) {
    json config = {
        {"loss", {
            {"otype", "RelativeL2Luminance"}
        }},
        {"optimizer", {
            {"otype", "EMA"},
            {"decay", 0.99f},
            {"nesteed", {
                {"otype", "Adam"},
                {"learning_rate", learningRate},
                {"beta1", 0.9f},
                {"beta2", 0.99f},
                {"l2_reg", 0.0f}
            }}
        }},
        {"encoding", {
            {"otype", "NRC"}
        }},
        {"network", {
            {"otype", "FullyFusedMLP"},
            {"n_neurons", 64},
            {"n_hidden_layers", numHiddenLayers},
            {"activation", "ReLU"},
            {"output_activation", "None"},
        }}
    };

    m->loss.reset(create_loss<precision_t>(config.value("loss", json::object())));
    m->optimizer.reset(create_optimizer<precision_t>(config.value("optimizer", json::object())));
    m->network = std::make_shared<NetworkWithInputEncoding<precision_t>>(
        numInputDims, numOutputDims,
        config.value("encoding", json::object()),
        config.value("network", json::object()));

    m->trainer = std::make_shared<Trainer<float, precision_t, precision_t>>(
        m->network, m->optimizer, m->loss);
}

void NeuralRadianceCache::finalize() {
    m->trainer = nullptr;
    m->network = nullptr;
    m->optimizer = nullptr;
    m->loss = nullptr;
}

void NeuralRadianceCache::infer(
    hipStream_t stream, float* inputData, uint32_t numData, float* predictionData) {
    Assert((numData & 0x7F) == 0, "numData must be a multiple of 128.");
    GPUMatrix<float> inputs(inputData, numInputDims, numData);
    GPUMatrix<float> predictions(predictionData, numOutputDims, numData);
    m->network->inference(stream, inputs, predictions);
}

void NeuralRadianceCache::train(
    hipStream_t stream, float* inputData, float* targetData, uint32_t numData) {
    Assert((numData & 0x7F) == 0, "numData must be a multiple of 128.");
    GPUMatrix<float> inputs(inputData, numInputDims, numData);
    GPUMatrix<float> targets(targetData, numOutputDims, numData);
    m->trainer->training_step(stream, inputs, targets, nullptr);
}
