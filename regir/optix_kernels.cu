#include "hip/hip_runtime.h"
﻿#include "regir_shared.h"

using namespace shared;

struct HitPointParameter {
    float b1, b2;
    int32_t primIndex;

    CUDA_DEVICE_FUNCTION static HitPointParameter get() {
        HitPointParameter ret;
        float2 bc = optixGetTriangleBarycentrics();
        ret.b1 = bc.x;
        ret.b2 = bc.y;
        ret.primIndex = optixGetPrimitiveIndex();
        return ret;
    }
};

struct HitGroupSBTRecordData {
    GeometryInstanceData geomInstData;

    CUDA_DEVICE_FUNCTION static const HitGroupSBTRecordData &get() {
        return *reinterpret_cast<HitGroupSBTRecordData*>(optixGetSbtDataPointer());
    }
};



CUDA_DEVICE_KERNEL void RT_AH_NAME(visibility)() {
    float visibility = 0.0f;
    optixu::setPayloads<VisibilityRayPayloadSignature>(&visibility);
}



CUDA_DEVICE_FUNCTION uint32_t calcCellLinearIndex(const float3 &positionInWorld) {
    float3 relPos = positionInWorld - plp.s->gridOrigin;
    uint32_t ix = min(max(static_cast<uint32_t>(relPos.x / plp.s->gridCellSize.x), 0u),
                      plp.s->gridDimension.x - 1);
    uint32_t iy = min(max(static_cast<uint32_t>(relPos.y / plp.s->gridCellSize.y), 0u),
                      plp.s->gridDimension.y - 1);
    uint32_t iz = min(max(static_cast<uint32_t>(relPos.z / plp.s->gridCellSize.z), 0u),
                      plp.s->gridDimension.z - 1);
    return iz * plp.s->gridDimension.x * plp.s->gridDimension.y
        + iy * plp.s->gridDimension.x
        + ix;
}



CUDA_DEVICE_KERNEL void RT_RG_NAME(setupGBuffers)() {
    uint2 launchIndex = make_uint2(optixGetLaunchIndex().x, optixGetLaunchIndex().y);

    const PerspectiveCamera &camera = plp.f->camera;
    float jx = 0.5f;
    float jy = 0.5f;
    if (plp.f->enableJittering) {
        // JP: ジッターをかけると現状の実装ではUnbiased要件を満たさないかもしれない。要検討。
        // EN: Jittering may break the requirements for unbiasedness with the current implementation.
        //     Need more consideration.
        PCG32RNG rng = plp.s->rngBuffer.read(launchIndex);
        jx = rng.getFloat0cTo1o();
        jy = rng.getFloat0cTo1o();
        plp.s->rngBuffer.write(launchIndex, rng);
    }
    float x = (launchIndex.x + jx) / plp.s->imageSize.x;
    float y = (launchIndex.y + jy) / plp.s->imageSize.y;
    float vh = 2 * std::tan(camera.fovY * 0.5f);
    float vw = camera.aspect * vh;

    float3 origin = camera.position;
    float3 direction = normalize(camera.orientation * make_float3(vw * (0.5f - x), vh * (0.5f - y), 1));

    HitPointParams hitPointParams;
    hitPointParams.positionInWorld = make_float3(NAN);
    hitPointParams.prevPositionInWorld = make_float3(NAN);
    hitPointParams.normalInWorld = make_float3(NAN);
    hitPointParams.texCoord = make_float2(NAN);
    hitPointParams.materialSlot = 0xFFFFFFFF;

    PickInfo pickInfo = {};

    HitPointParams* hitPointParamsPtr = &hitPointParams;
    PickInfo* pickInfoPtr = &pickInfo;
    optixu::trace<PrimaryRayPayloadSignature>(
        plp.f->travHandle, origin, direction,
        0.0f, FLT_MAX, 0.0f, 0xFF, OPTIX_RAY_FLAG_NONE,
        RayType_Primary, NumRayTypes, RayType_Primary,
        hitPointParamsPtr, pickInfoPtr);



    float2 curRasterPos = make_float2(launchIndex.x + 0.5f, launchIndex.y + 0.5f);
    float2 prevRasterPos =
        plp.f->prevCamera.calcScreenPosition(hitPointParams.prevPositionInWorld)
        * make_float2(plp.s->imageSize.x, plp.s->imageSize.y);
    float2 motionVector = curRasterPos - prevRasterPos;
    if (plp.f->resetFlowBuffer || isnan(hitPointParams.prevPositionInWorld.x))
        motionVector = make_float2(0.0f, 0.0f);

    GBuffer0 gBuffer0;
    gBuffer0.positionInWorld = hitPointParams.positionInWorld;
    gBuffer0.texCoord_x = hitPointParams.texCoord.x;
    GBuffer1 gBuffer1;
    gBuffer1.normalInWorld = hitPointParams.normalInWorld;
    gBuffer1.texCoord_y = hitPointParams.texCoord.y;
    GBuffer2 gBuffer2;
    gBuffer2.motionVector = motionVector;
    gBuffer2.materialSlot = hitPointParams.materialSlot;

    uint32_t bufIdx = plp.f->bufferIndex;
    plp.s->GBuffer0[bufIdx].write(launchIndex, gBuffer0);
    plp.s->GBuffer1[bufIdx].write(launchIndex, gBuffer1);
    plp.s->GBuffer2[bufIdx].write(launchIndex, gBuffer2);

    if (launchIndex.x == plp.f->mousePosition.x &&
        launchIndex.y == plp.f->mousePosition.y)
        *plp.f->pickInfo = pickInfo;

    // JP: デノイザーに必要な情報を出力。
    // EN: Output information required for the denoiser.
    float3 firstHitNormal = transpose(camera.orientation) * hitPointParams.normalInWorld;
    firstHitNormal.x *= -1;
    float3 prevAlbedoResult = make_float3(0.0f, 0.0f, 0.0f);
    float3 prevNormalResult = make_float3(0.0f, 0.0f, 0.0f);
    if (plp.f->numAccumFrames > 0) {
        prevAlbedoResult = getXYZ(plp.s->albedoAccumBuffer.read(launchIndex));
        prevNormalResult = getXYZ(plp.s->normalAccumBuffer.read(launchIndex));
    }
    float curWeight = 1.0f / (1 + plp.f->numAccumFrames);
    float3 albedoResult = (1 - curWeight) * prevAlbedoResult + curWeight * hitPointParams.albedo;
    float3 normalResult = (1 - curWeight) * prevNormalResult + curWeight * firstHitNormal;
    plp.s->albedoAccumBuffer.write(launchIndex, make_float4(albedoResult, 1.0f));
    plp.s->normalAccumBuffer.write(launchIndex, make_float4(normalResult, 1.0f));
}

CUDA_DEVICE_KERNEL void RT_CH_NAME(setupGBuffers)() {
    uint2 launchIndex = make_uint2(optixGetLaunchIndex().x, optixGetLaunchIndex().y);

    auto sbtr = HitGroupSBTRecordData::get();
    const InstanceData &inst = plp.f->instanceDataBuffer[optixGetInstanceId()];
    const GeometryInstanceData &geomInst = sbtr.geomInstData;

    HitPointParams* hitPointParams;
    PickInfo* pickInfo;
    optixu::getPayloads<PrimaryRayPayloadSignature>(&hitPointParams, &pickInfo);

    auto hp = HitPointParameter::get();
    float3 positionInWorld;
    float3 prevPositionInWorld;
    float3 shadingNormalInWorld;
    float3 texCoord0DirInWorld;
    //float3 geometricNormalInWorld;
    float2 texCoord;
    {
        const Triangle &tri = geomInst.triangleBuffer[hp.primIndex];
        const Vertex &v0 = geomInst.vertexBuffer[tri.index0];
        const Vertex &v1 = geomInst.vertexBuffer[tri.index1];
        const Vertex &v2 = geomInst.vertexBuffer[tri.index2];
        float b1 = hp.b1;
        float b2 = hp.b2;
        float b0 = 1 - (b1 + b2);
        float3 localP = b0 * v0.position + b1 * v1.position + b2 * v2.position;
        shadingNormalInWorld = b0 * v0.normal + b1 * v1.normal + b2 * v2.normal;
        texCoord0DirInWorld = b0 * v0.texCoord0Dir + b1 * v1.texCoord0Dir + b2 * v2.texCoord0Dir;
        //geometricNormalInWorld = cross(v1.position - v0.position, v2.position - v0.position);
        texCoord = b0 * v0.texCoord + b1 * v1.texCoord + b2 * v2.texCoord;

        positionInWorld = optixTransformPointFromObjectToWorldSpace(localP);
        prevPositionInWorld = inst.prevTransform * localP;
        shadingNormalInWorld = normalize(optixTransformNormalFromObjectToWorldSpace(shadingNormalInWorld));
        texCoord0DirInWorld = normalize(optixTransformVectorFromObjectToWorldSpace(texCoord0DirInWorld));
        //geometricNormalInWorld = normalize(optixTransformNormalFromObjectToWorldSpace(geometricNormalInWorld));
        if (!allFinite(shadingNormalInWorld)) {
            shadingNormalInWorld = make_float3(0, 0, 1);
            texCoord0DirInWorld = make_float3(1, 0, 0);
        }
    }

    const MaterialData &mat = plp.s->materialDataBuffer[geomInst.materialSlot];

    BSDF bsdf;
    bsdf.setup(mat, texCoord);
    ReferenceFrame shadingFrame(shadingNormalInWorld, texCoord0DirInWorld);
    float3 modLocalNormal = mat.readModifiedNormal(mat.normal, texCoord, mat.normalDimension);
    if (plp.f->enableBumpMapping)
        applyBumpMapping(modLocalNormal, &shadingFrame);
    float3 vOut = -optixGetWorldRayDirection();
    float3 vOutLocal = shadingFrame.toLocal(normalize(vOut));

    hitPointParams->albedo = bsdf.evaluateDHReflectanceEstimate(vOutLocal);
    hitPointParams->positionInWorld = positionInWorld;
    hitPointParams->prevPositionInWorld = prevPositionInWorld;
    hitPointParams->normalInWorld = shadingFrame.normal;
    hitPointParams->texCoord = texCoord;
    hitPointParams->materialSlot = geomInst.materialSlot;

    // JP: マウスが乗っているピクセルの情報を出力する。
    // EN: Export the information of the pixel on which the mouse is.
    if (launchIndex.x == plp.f->mousePosition.x &&
        launchIndex.y == plp.f->mousePosition.y) {
        pickInfo->hit = true;
        pickInfo->instSlot = optixGetInstanceId();
        pickInfo->geomInstSlot = geomInst.geomInstSlot;
        pickInfo->matSlot = geomInst.materialSlot;
        pickInfo->primIndex = hp.primIndex;
        pickInfo->positionInWorld = positionInWorld;
        pickInfo->normalInWorld = shadingFrame.normal;
        pickInfo->albedo = hitPointParams->albedo;
        float3 emittance = make_float3(0.0f, 0.0f, 0.0f);
        if (mat.emittance) {
            float4 texValue = tex2DLod<float4>(mat.emittance, texCoord.x, texCoord.y, 0.0f);
            emittance = make_float3(texValue);
        }
        pickInfo->emittance = emittance;
        pickInfo->cellLinearIndex = calcCellLinearIndex(positionInWorld);
    }
}

CUDA_DEVICE_KERNEL void RT_MS_NAME(setupGBuffers)() {
    uint2 launchIndex = make_uint2(optixGetLaunchIndex().x, optixGetLaunchIndex().y);

    float3 vOut = -optixGetWorldRayDirection();
    float3 p = -vOut;

    float posPhi, posTheta;
    toPolarYUp(p, &posPhi, &posTheta);

    float phi = posPhi + plp.f->envLightRotation;

    float u = phi / (2 * Pi);
    u -= floorf(u);
    float v = posTheta / Pi;

    HitPointParams* hitPointParams;
    PickInfo* pickInfo;
    optixu::getPayloads<PrimaryRayPayloadSignature>(&hitPointParams, &pickInfo);

    hitPointParams->albedo = make_float3(0.0f, 0.0f, 0.0f);
    hitPointParams->positionInWorld = p;
    hitPointParams->prevPositionInWorld = p;
    hitPointParams->normalInWorld = vOut;
    hitPointParams->texCoord = make_float2(u, v);
    hitPointParams->materialSlot = 0xFFFFFFFF;

    // JP: マウスが乗っているピクセルの情報を出力する。
    // EN: Export the information of the pixel on which the mouse is.
    if (launchIndex.x == plp.f->mousePosition.x &&
        launchIndex.y == plp.f->mousePosition.y) {
        pickInfo->hit = true;
        pickInfo->instSlot = 0xFFFFFFFF;
        pickInfo->geomInstSlot = 0xFFFFFFFF;
        pickInfo->matSlot = 0xFFFFFFFF;
        pickInfo->primIndex = 0xFFFFFFFF;
        pickInfo->positionInWorld = p;
        pickInfo->albedo = make_float3(0.0f, 0.0f, 0.0f);
        float3 emittance = make_float3(0.0f, 0.0f, 0.0f);
        if (plp.s->envLightTexture && plp.f->enableEnvLight) {
            float4 texValue = tex2DLod<float4>(plp.s->envLightTexture, u, v, 0.0f);
            emittance = make_float3(texValue);
            emittance *= Pi * plp.f->envLightPowerCoeff;
        }
        pickInfo->emittance = emittance;
        pickInfo->normalInWorld = vOut;
    }
}



static constexpr bool useImplicitLightSampling = true;
static constexpr bool useExplicitLightSampling = true;
static constexpr bool useMultipleImportanceSampling = useImplicitLightSampling && useExplicitLightSampling;
static_assert(useImplicitLightSampling || useExplicitLightSampling, "Invalid configuration for light sampling.");

CUDA_DEVICE_FUNCTION float3 sampleFromCell(
    const float3 &shadingPoint, const float3 &vOutLocal, const ReferenceFrame &shadingFrame, const BSDF &bsdf,
    uint32_t frameIndex, PCG32RNG &rng,
    LightSample* lightSample, float* recProbDensityEstimate) {
    float3 randomOffset;
    if (plp.f->enableCellRandomization) {
        randomOffset = plp.s->gridCellSize
            * make_float3(-0.5f + rng.getFloat0cTo1o(),
                          -0.5f + rng.getFloat0cTo1o(),
                          -0.5f + rng.getFloat0cTo1o());
    }
    else {
        randomOffset = make_float3(0.0f);
    }
    uint32_t cellLinearIndex = calcCellLinearIndex(shadingPoint + randomOffset);
    uint32_t resStartIndex = kNumLightSlotsPerCell * cellLinearIndex;

    // JP: セルに触れたフラグを建てておく。
    // EN: Set the flag indicating the cell is touched.
    atomicAdd(&plp.s->perCellNumAccesses[cellLinearIndex], 1u);

    // JP: セルごとに保持している複数のReservoirからリサンプリングを行う。
    // EN: Resample from multiple reservoirs held by each cell.
    const uint32_t numResampling = 1 << plp.f->log2NumCandidatesPerCell;
    Reservoir<LightSample> combinedReservoir;
    combinedReservoir.initialize();
    uint32_t combinedStreamLength = 0;
    float3 selectedContribution = make_float3(0.0f);
    float selectedTargetPDensity = 0.0f;
    for (int i = 0; i < numResampling; ++i) {
        uint32_t lightSlotIdx = resStartIndex + mapPrimarySampleToDiscrete(rng.getFloat0cTo1o(), kNumLightSlotsPerCell);
        const Reservoir<LightSample> &r = plp.s->reservoirs[plp.f->bufferIndex][lightSlotIdx];
        const ReservoirInfo &rInfo = plp.s->reservoirInfos[plp.f->bufferIndex][lightSlotIdx];
        const LightSample &lightSample = r.getSample();
        uint32_t streamLength = r.getStreamLength();
        combinedStreamLength += streamLength;
        if (rInfo.recPDFEstimate == 0.0f)
            continue;

        // JP: Unshadowed ContributionをターゲットPDFとする。
        // EN: Use unshadowed constribution as the target PDF.
        float3 cont = performDirectLighting<false>(shadingPoint, vOutLocal, shadingFrame, bsdf, lightSample);
        float targetPDensity = convertToWeight(cont);

        // JP: ソースのターゲットPDFとここでのターゲットPDFは異なるためサンプルにはウェイトがかかる。
        // EN: The sample has a weight since the source PDF and the target PDF hre are different.
        float weight = targetPDensity * rInfo.recPDFEstimate * streamLength;
        if (combinedReservoir.update(lightSample, weight, rng.getFloat0cTo1o())) {
            selectedContribution = cont;
            selectedTargetPDensity = targetPDensity;
        }
    }
    combinedReservoir.setStreamLength(combinedStreamLength);

    *lightSample = combinedReservoir.getSample();

    float weightForEstimate = 1.0f / combinedReservoir.getStreamLength();
    *recProbDensityEstimate = weightForEstimate * combinedReservoir.getSumWeights() / selectedTargetPDensity;
    if (!isfinite(*recProbDensityEstimate))
        *recProbDensityEstimate = 0.0f;

    return selectedContribution;
}

template <bool useReGIR>
CUDA_DEVICE_FUNCTION float3 performNextEventEstimation(
    const float3 &shadingPoint, const float3 &vOutLocal, const ReferenceFrame &shadingFrame, const BSDF &bsdf,
    PCG32RNG &rng) {
    float3 ret = make_float3(0.0f);
    if constexpr (useReGIR) {
        LightSample lightSample;
        float recProbDensityEstimate;
        float3 unshadowedContribution = sampleFromCell(
            shadingPoint, vOutLocal, shadingFrame, bsdf,
            plp.f->frameIndex, rng,
            &lightSample, &recProbDensityEstimate);
        if (recProbDensityEstimate > 0.0f) {
            float visibility = evaluateVisibility(shadingPoint, lightSample);
            ret = unshadowedContribution * (visibility * recProbDensityEstimate);
        }
    }
    else {
        if constexpr (useExplicitLightSampling) {
            float uLight = rng.getFloat0cTo1o();
            bool selectEnvLight = false;
            float probToSampleCurLightType = 1.0f;
            if (plp.s->envLightTexture && plp.f->enableEnvLight) {
                if (uLight < probToSampleEnvLight) {
                    probToSampleCurLightType = probToSampleEnvLight;
                    uLight /= probToSampleCurLightType;
                    selectEnvLight = true;
                }
                else {
                    probToSampleCurLightType = 1.0f - probToSampleEnvLight;
                    uLight = (uLight - probToSampleEnvLight) / probToSampleCurLightType;
                }
            }
            LightSample lightSample;
            float areaPDensity;
            sampleLight(uLight, selectEnvLight, rng.getFloat0cTo1o(), rng.getFloat0cTo1o(),
                        &lightSample, &areaPDensity);
            areaPDensity *= probToSampleCurLightType;
            float misWeight = 1.0f;
            if constexpr (useMultipleImportanceSampling) {
                float3 shadowRay = lightSample.atInfinity ?
                    lightSample.position :
                    (lightSample.position - shadingPoint);
                float dist2 = sqLength(shadowRay);
                shadowRay /= std::sqrt(dist2);
                float3 vInLocal = shadingFrame.toLocal(shadowRay);
                float lpCos = std::fabs(dot(shadowRay, lightSample.normal));
                float bsdfPDensity = bsdf.evaluatePDF(vOutLocal, vInLocal) * lpCos / dist2;
                if (!isfinite(bsdfPDensity))
                    bsdfPDensity = 0.0f;
                float lightPDensity = areaPDensity;
                misWeight = pow2(lightPDensity) / (pow2(bsdfPDensity) + pow2(lightPDensity));
            }
            ret = performDirectLighting<true>(
                shadingPoint, vOutLocal, shadingFrame, bsdf, lightSample) * (misWeight / areaPDensity);
        }
    }

    return ret;
}

template <bool useReGIR>
CUDA_DEVICE_FUNCTION void pathTrace_rayGen_generic() {
    uint2 launchIndex = make_uint2(optixGetLaunchIndex().x, optixGetLaunchIndex().y);

    uint32_t bufIdx = plp.f->bufferIndex;
    GBuffer0 gBuffer0 = plp.s->GBuffer0[bufIdx].read(launchIndex);
    GBuffer1 gBuffer1 = plp.s->GBuffer1[bufIdx].read(launchIndex);
    GBuffer2 gBuffer2 = plp.s->GBuffer2[bufIdx].read(launchIndex);

    float3 positionInWorld = gBuffer0.positionInWorld;
    float3 shadingNormalInWorld = gBuffer1.normalInWorld;
    float2 texCoord = make_float2(gBuffer0.texCoord_x, gBuffer1.texCoord_y);
    uint32_t materialSlot = gBuffer2.materialSlot;

    const PerspectiveCamera &camera = plp.f->camera;

    bool useEnvLight = plp.s->envLightTexture && plp.f->enableEnvLight;
    float3 contribution = make_float3(0.01f, 0.01f, 0.01f);
    if (materialSlot != 0xFFFFFFFF) {
        float3 alpha = make_float3(1.0f);
        float initImportance = sRGB_calcLuminance(alpha);
        PCG32RNG rng = plp.s->rngBuffer.read(launchIndex);

        // JP: 最初の交点におけるシェーディング。
        // EN: Shading on the first hit.
        float3 vIn;
        float dirPDensity;
        {
            const MaterialData &mat = plp.s->materialDataBuffer[materialSlot];

            // TODO?: Use true geometric normal.
            float3 geometricNormalInWorld = shadingNormalInWorld;
            float3 vOut = normalize(camera.position - positionInWorld);
            float frontHit = dot(vOut, geometricNormalInWorld) >= 0.0f ? 1.0f : -1.0f;

            ReferenceFrame shadingFrame(shadingNormalInWorld);
            positionInWorld = offsetRayOriginNaive(positionInWorld, frontHit * geometricNormalInWorld);
            float3 vOutLocal = shadingFrame.toLocal(vOut);

            // JP: 光源を直接見ている場合の寄与を蓄積。
            // EN: Accumulate the contribution from a light source directly seeing.
            contribution = make_float3(0.0f);
            if (vOutLocal.z > 0 && mat.emittance) {
                float4 texValue = tex2DLod<float4>(mat.emittance, texCoord.x, texCoord.y, 0.0f);
                float3 emittance = make_float3(texValue);
                contribution += alpha * emittance / Pi;
            }

            BSDF bsdf;
            bsdf.setup(mat, texCoord);

            // Next event estimation (explicit light sampling) on the first hit.
            contribution += alpha * performNextEventEstimation<useReGIR>(
                positionInWorld, vOutLocal, shadingFrame, bsdf, rng);

            // generate a next ray.
            float3 vInLocal;
            alpha *= bsdf.sampleThroughput(
                vOutLocal, rng.getFloat0cTo1o(), rng.getFloat0cTo1o(),
                &vInLocal, &dirPDensity);
            vIn = shadingFrame.fromLocal(vInLocal);
        }

        // Path extension loop
        PathTraceWriteOnlyPayload woPayload = {};
        PathTraceWriteOnlyPayload* woPayloadPtr = &woPayload;
        PathTraceReadWritePayload rwPayload = {};
        PathTraceReadWritePayload* rwPayloadPtr = &rwPayload;
        rwPayload.rng = rng;
        rwPayload.initImportance = initImportance;
        rwPayload.alpha = alpha;
        rwPayload.prevDirPDensity = dirPDensity;
        rwPayload.contribution = contribution;
        rwPayload.pathLength = 1;
        float3 rayOrg = positionInWorld;
        float3 rayDir = vIn;
        while (true) {
            bool isValidSampling = rwPayload.prevDirPDensity > 0.0f && isfinite(rwPayload.prevDirPDensity);
            if (!isValidSampling)
                break;

            ++rwPayload.pathLength;
            if (rwPayload.pathLength >= plp.f->maxPathLength)
                rwPayload.maxLengthTerminate = true;
            rwPayload.terminate = true;
            // JP: 経路長制限に到達したときに、implicit light samplingを使わない場合はClosest-hit program内
            //     で行うことが無いので終了する。
            // EN: Nothing to do in the closest-hit program when reaching the path length limit
            //     in the case implicit light sampling is unused.
            if constexpr (useReGIR || !useImplicitLightSampling) {
                if (rwPayload.maxLengthTerminate)
                    break;
                // Russian roulette
                float continueProb = std::fmin(sRGB_calcLuminance(rwPayload.alpha) / rwPayload.initImportance, 1.0f);
                if (rwPayload.rng.getFloat0cTo1o() >= continueProb)
                    break;
                rwPayload.alpha /= continueProb;
            }

            constexpr RayType pathTraceRayType = useReGIR ? RayType_PathTraceReGIR : RayType_PathTraceBaseline;
            optixu::trace<PathTraceRayPayloadSignature>(
                plp.f->travHandle, rayOrg, rayDir,
                0.0f, FLT_MAX, 0.0f, 0xFF, OPTIX_RAY_FLAG_NONE,
                pathTraceRayType, NumRayTypes, pathTraceRayType,
                woPayloadPtr, rwPayloadPtr);
            if (rwPayload.terminate)
                break;
            rayOrg = woPayload.nextOrigin;
            rayDir = woPayload.nextDirection;
        }
        contribution = rwPayload.contribution;

        plp.s->rngBuffer.write(launchIndex, rwPayload.rng);
    }
    else {
        // JP: 環境光源を直接見ている場合の寄与を蓄積。
        // EN: Accumulate the contribution from the environmental light source directly seeing.
        if (useEnvLight) {
            float u = texCoord.x, v = texCoord.y;
            float4 texValue = tex2DLod<float4>(plp.s->envLightTexture, u, v, 0.0f);
            float3 luminance = plp.f->envLightPowerCoeff * make_float3(texValue);
            contribution = luminance;
        }
    }

    float3 prevColorResult = make_float3(0.0f, 0.0f, 0.0f);
    if (plp.f->numAccumFrames > 0)
        prevColorResult = getXYZ(plp.s->beautyAccumBuffer.read(launchIndex));
    float curWeight = 1.0f / (1 + plp.f->numAccumFrames);
    float3 colorResult = (1 - curWeight) * prevColorResult + curWeight * contribution;
    plp.s->beautyAccumBuffer.write(launchIndex, make_float4(colorResult, 1.0f));
}

template <bool useReGIR>
CUDA_DEVICE_FUNCTION void pathTrace_closestHit_generic() {
    auto sbtr = HitGroupSBTRecordData::get();
    const InstanceData &inst = plp.f->instanceDataBuffer[optixGetInstanceId()];
    const GeometryInstanceData &geomInst = sbtr.geomInstData;

    PathTraceWriteOnlyPayload* woPayload;
    PathTraceReadWritePayload* rwPayload;
    optixu::getPayloads<PathTraceRayPayloadSignature>(&woPayload, &rwPayload);
    PCG32RNG &rng = rwPayload->rng;

    auto hp = HitPointParameter::get();
    float3 positionInWorld;
    float3 shadingNormalInWorld;
    float3 texCoord0DirInWorld;
    float3 geometricNormalInWorld;
    float2 texCoord;
    float hypAreaPDensity;
    {
        const Triangle &tri = geomInst.triangleBuffer[hp.primIndex];
        const Vertex &v0 = geomInst.vertexBuffer[tri.index0];
        const Vertex &v1 = geomInst.vertexBuffer[tri.index1];
        const Vertex &v2 = geomInst.vertexBuffer[tri.index2];
        float b1 = hp.b1;
        float b2 = hp.b2;
        float b0 = 1 - (b1 + b2);
        float3 localP = b0 * v0.position + b1 * v1.position + b2 * v2.position;
        shadingNormalInWorld = b0 * v0.normal + b1 * v1.normal + b2 * v2.normal;
        texCoord0DirInWorld = b0 * v0.texCoord0Dir + b1 * v1.texCoord0Dir + b2 * v2.texCoord0Dir;
        geometricNormalInWorld = cross(v1.position - v0.position, v2.position - v0.position);
        if constexpr (useMultipleImportanceSampling && !useReGIR) {
            // JP: 交点をExplicit Light Samplingでサンプルする場合の仮想的な確率密度を求める。
            // EN: Compute a hypothetical probability density with which the intersection point
            //     is sampled by explicit light sampling.
            float lightProb = 1.0f;
            if (plp.s->envLightTexture && plp.f->enableEnvLight)
                lightProb *= (1 - probToSampleEnvLight);
            lightProb *= inst.lightGeomInstDist.integral() / plp.s->lightInstDist.integral();
            lightProb *= geomInst.emitterPrimDist.integral() / inst.lightGeomInstDist.integral();
            lightProb *= geomInst.emitterPrimDist.evaluatePMF(hp.primIndex);
            float area = 0.5f * length(geometricNormalInWorld);
            hypAreaPDensity = lightProb / area;
        }
        texCoord = b0 * v0.texCoord + b1 * v1.texCoord + b2 * v2.texCoord;

        positionInWorld = optixTransformPointFromObjectToWorldSpace(localP);
        shadingNormalInWorld = normalize(optixTransformNormalFromObjectToWorldSpace(shadingNormalInWorld));
        texCoord0DirInWorld = normalize(optixTransformVectorFromObjectToWorldSpace(texCoord0DirInWorld));
        geometricNormalInWorld = normalize(optixTransformNormalFromObjectToWorldSpace(geometricNormalInWorld));
        if (!allFinite(shadingNormalInWorld)) {
            shadingNormalInWorld = make_float3(0, 0, 1);
            texCoord0DirInWorld = make_float3(1, 0, 0);
        }
    }

    const MaterialData &mat = plp.s->materialDataBuffer[geomInst.materialSlot];

    float3 vOut = normalize(-optixGetWorldRayDirection());
    float frontHit = dot(vOut, geometricNormalInWorld) >= 0.0f ? 1.0f : -1.0f;

    ReferenceFrame shadingFrame(shadingNormalInWorld, texCoord0DirInWorld);
    float3 modLocalNormal = mat.readModifiedNormal(mat.normal, texCoord, mat.normalDimension);
    if (plp.f->enableBumpMapping)
        applyBumpMapping(modLocalNormal, &shadingFrame);
    positionInWorld = offsetRayOrigin(positionInWorld, frontHit * geometricNormalInWorld);
    float3 vOutLocal = shadingFrame.toLocal(vOut);

    if constexpr (useImplicitLightSampling && !useReGIR) {
        // Implicit Light Sampling
        if (vOutLocal.z > 0 && mat.emittance) {
            float4 texValue = tex2DLod<float4>(mat.emittance, texCoord.x, texCoord.y, 0.0f);
            float3 emittance = make_float3(texValue);
            float misWeight = 1.0f;
            if constexpr (useMultipleImportanceSampling) {
                float dist2 = squaredDistance(optixGetWorldRayOrigin(), positionInWorld);
                float lightPDensity = hypAreaPDensity * dist2 / vOutLocal.z;
                float bsdfPDensity = rwPayload->prevDirPDensity;
                misWeight = pow2(bsdfPDensity) / (pow2(bsdfPDensity) + pow2(lightPDensity));
            }
            rwPayload->contribution += rwPayload->alpha * emittance * (misWeight / Pi);
        }

        // Russian roulette
        float continueProb = std::fmin(sRGB_calcLuminance(rwPayload->alpha) / rwPayload->initImportance, 1.0f);
        if (rng.getFloat0cTo1o() >= continueProb || rwPayload->maxLengthTerminate)
            return;
        rwPayload->alpha /= continueProb;
    }

    BSDF bsdf;
    bsdf.setup(mat, texCoord);

    // Next Event Estimation (Explicit Light Sampling)
    rwPayload->contribution += rwPayload->alpha * performNextEventEstimation<useReGIR>(
        positionInWorld, vOutLocal, shadingFrame, bsdf, rng);

    // generate a next ray.
    float3 vInLocal;
    float dirPDensity;
    rwPayload->alpha *= bsdf.sampleThroughput(
        vOutLocal, rng.getFloat0cTo1o(), rng.getFloat0cTo1o(),
        &vInLocal, &dirPDensity);
    float3 vIn = shadingFrame.fromLocal(vInLocal);

    woPayload->nextOrigin = positionInWorld;
    woPayload->nextDirection = vIn;
    rwPayload->prevDirPDensity = dirPDensity;
    rwPayload->terminate = false;
}

CUDA_DEVICE_KERNEL void RT_RG_NAME(pathTraceBaseline)() {
    pathTrace_rayGen_generic<false>();
}

CUDA_DEVICE_KERNEL void RT_CH_NAME(pathTraceBaseline)() {
    pathTrace_closestHit_generic<false>();
}

CUDA_DEVICE_KERNEL void RT_MS_NAME(pathTraceBaseline)() {
    if constexpr (useImplicitLightSampling) {
        if (!plp.s->envLightTexture || !plp.f->enableEnvLight)
            return;

        PathTraceReadWritePayload* rwPayload;
        optixu::getPayloads<PathTraceRayPayloadSignature>(nullptr, &rwPayload);

        float3 rayDir = normalize(optixGetWorldRayDirection());
        float posPhi, theta;
        toPolarYUp(rayDir, &posPhi, &theta);

        float phi = posPhi + plp.f->envLightRotation;
        phi = phi - floorf(phi / (2 * Pi)) * 2 * Pi;
        float2 texCoord = make_float2(phi / (2 * Pi), theta / Pi);

        // Implicit Light Sampling
        float4 texValue = tex2DLod<float4>(plp.s->envLightTexture, texCoord.x, texCoord.y, 0.0f);
        float3 luminance = plp.f->envLightPowerCoeff * make_float3(texValue);
        float misWeight = 1.0f;
        if constexpr (useMultipleImportanceSampling) {
            float uvPDF = plp.s->envLightImportanceMap.evaluatePDF(texCoord.x, texCoord.y);
            float hypAreaPDensity = uvPDF / (2 * Pi * Pi * std::sin(theta));
            float lightPDensity = probToSampleEnvLight * hypAreaPDensity;
            float bsdfPDensity = rwPayload->prevDirPDensity;
            misWeight = pow2(bsdfPDensity) / (pow2(bsdfPDensity) + pow2(lightPDensity));
        }
        rwPayload->contribution += rwPayload->alpha * luminance * misWeight;
    }
}

CUDA_DEVICE_KERNEL void RT_RG_NAME(pathTraceRegir)() {
    pathTrace_rayGen_generic<true>();
}

CUDA_DEVICE_KERNEL void RT_CH_NAME(pathTraceRegir)() {
    pathTrace_closestHit_generic<true>();
}
