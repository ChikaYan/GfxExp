#include "hip/hip_runtime.h"
﻿#include "regir_shared.h"
#include "../common/common_device.cuh"

using namespace shared;

CUDA_DEVICE_MEM static PipelineLaunchParameters plp;

CUDA_DEVICE_FUNCTION void sampleLight(
    float ul, bool sampleEnvLight, float u0, float u1,
    LightSample* lightSample, float* areaPDensity) {
    hipTextureObject_t texEmittance = 0;
    float3 emittance = make_float3(0.0f, 0.0f, 0.0f);
    float2 texCoord;
    if (sampleEnvLight) {
        float u, v;
        float uvPDF;
        plp.s->envLightImportanceMap.sample(u0, u1, &u, &v, &uvPDF);
        float phi = 2 * Pi * u;
        float theta = Pi * v;

        float posPhi = phi - plp.f->envLightRotation;
        posPhi = posPhi - floorf(posPhi / (2 * Pi)) * 2 * Pi;

        float3 direction = fromPolarYUp(posPhi, theta);
        float3 position = make_float3(direction.x, direction.y, direction.z);
        lightSample->position = position;
        lightSample->atInfinity = true;

        lightSample->normal = -position;

        // JP: テクスチャー空間中のPDFを面積に関するものに変換する。
        // EN: convert the PDF in texture space to one with respect to area.
        // The true value is: lim_{l to inf} uvPDF / (2 * Pi * Pi * std::sin(theta)) / l^2
        *areaPDensity = uvPDF / (2 * Pi * Pi * std::sin(theta));

        texEmittance = plp.s->envLightTexture;
        // JP: 環境マップテクスチャーの値に係数をかけて、通常の光源と同じように返り値を光束発散度
        //     として扱えるようにする。
        // EN: Multiply a coefficient to make the return value possible to be handled as luminous emittance.
        emittance = make_float3(Pi * plp.f->envLightPowerCoeff);
        texCoord.x = u;
        texCoord.y = v;
    }
    else {
        float lightProb = 1.0f;

        // JP: まずはインスタンスをサンプルする。
        // EN: First, sample an instance.
        float instProb;
        float uGeomInst;
        uint32_t instIndex = plp.s->lightInstDist.sample(ul, &instProb, &uGeomInst);
        lightProb *= instProb;
        const InstanceData &inst = plp.f->instanceDataBuffer[instIndex];

        // JP: 次にサンプルしたインスタンスに属するジオメトリインスタンスをサンプルする。
        // EN: Next, sample a geometry instance which belongs to the sampled instance.
        float geomInstProb;
        float uPrim;
        uint32_t geomInstIndexInInst = inst.lightGeomInstDist.sample(uGeomInst, &geomInstProb, &uPrim);
        uint32_t geomInstIndex = inst.geomInstSlots[geomInstIndexInInst];
        lightProb *= geomInstProb;
        const GeometryInstanceData &geomInst = plp.s->geometryInstanceDataBuffer[geomInstIndex];

        // JP: 最後に、サンプルしたジオメトリインスタンスに属するプリミティブをサンプルする。
        // EN: Finally, sample a primitive which belongs to the sampled geometry instance.
        float primProb;
        uint32_t primIndex = geomInst.emitterPrimDist.sample(uPrim, &primProb);
        lightProb *= primProb;

        // Uniform sampling on unit triangle
        // A Low-Distortion Map Between Triangle and Square
        float t0 = 0.5f * u0;
        float t1 = 0.5f * u1;
        float offset = t1 - t0;
        if (offset > 0)
            t1 += offset;
        else
            t0 -= offset;
        float t2 = 1 - (t0 + t1);

        //printf("%u-%u-%u: %g\n", instIndex, geomInstIndex, primIndex, lightProb);

        const MaterialData &mat = plp.s->materialDataBuffer[geomInst.materialSlot];

        const shared::Triangle &tri = geomInst.triangleBuffer[primIndex];
        const shared::Vertex (&v)[3] = {
            geomInst.vertexBuffer[tri.index0],
            geomInst.vertexBuffer[tri.index1],
            geomInst.vertexBuffer[tri.index2]
        };
        float3 p[3] = {
            inst.transform * v[0].position,
            inst.transform * v[1].position,
            inst.transform * v[2].position,
        };

        float3 geomNormal = cross(p[1] - p[0], p[2] - p[0]);
        lightSample->position = t0 * p[0] + t1 * p[1] + t2 * p[2];
        lightSample->atInfinity = false;
        float recArea = 1.0f / length(geomNormal);
        //lightSample->normal = geomNormal * recArea;
        lightSample->normal = t0 * v[0].normal + t1 * v[1].normal + t2 * v[2].normal;
        lightSample->normal = normalize(inst.normalMatrix * lightSample->normal);
        recArea *= 2;
        *areaPDensity = lightProb * recArea;

        //printf("%u-%u-%u: (%g, %g, %g), PDF: %g\n", instIndex, geomInstIndex, primIndex,
        //       mat.emittance.x, mat.emittance.y, mat.emittance.z, *areaPDensity);

        //printf("%u-%u-%u: (%g, %g, %g), (%g, %g, %g)\n", instIndex, geomInstIndex, primIndex,
        //       lightPosition->x, lightPosition->y, lightPosition->z,
        //       lightNormal->x, lightNormal->y, lightNormal->z);

        if (mat.emittance) {
            texEmittance = mat.emittance;
            emittance = make_float3(1.0f, 1.0f, 1.0f);
            texCoord = t0 * v[0].texCoord + t1 * v[1].texCoord + t2 * v[2].texCoord;
        }
    }

    if (texEmittance) {
        float4 texValue = tex2DLod<float4>(texEmittance, texCoord.x, texCoord.y, 0.0f);
        emittance *= make_float3(texValue);
    }
    lightSample->emittance = emittance;
}

// TODO: セルの中央だけのサンプリングだと、セルの中央が光源の裏側に回ってしまっている場合に、
//       寄与の可能性のあるサンプルを棄却してしまう。代表点をランダムに決定するなどで解決できそうだが、
//       PDFが毎回変わるのでそれを考慮する必要あり？
CUDA_DEVICE_FUNCTION float3 sampleIntensity(
    const float3 &shadingPoint, float minSquaredDistance,
    float uLight, bool sampleEnvLight, float uPos0, float uPos1,
    LightSample* lightSample, float* probDensity) {
    sampleLight(uLight, sampleEnvLight, uPos0, uPos1,
                lightSample, probDensity);

    float3 shadowRayDir = lightSample->atInfinity ?
        lightSample->position :
        (lightSample->position - shadingPoint);
    float dist2 = sqLength(shadowRayDir);
    float dist = std::sqrt(dist2);
    shadowRayDir /= dist;

    float lpCos = dot(-shadowRayDir, lightSample->normal);

    if (lpCos > 0) {
        float3 Le = lightSample->emittance / Pi;
        float3 ret = Le * (lpCos / dist2);
        return ret;
    }
    else {
        return make_float3(0.0f, 0.0f, 0.0f);
    }
}

template <bool useTemporalReuse>
CUDA_DEVICE_FUNCTION void buildCellReservoirsAndTemporalReuse(const PipelineLaunchParameters &_plp, uint32_t frameIndex) {
    plp = _plp;

    uint32_t linearThreadIndex = blockDim.x * blockIdx.x + threadIdx.x;
    uint32_t cellLinearIndex = linearThreadIndex / kNumLightSlotsPerCell;
    uint32_t lastAccessFrameIndex = plp.s->lastAccessFrameIndices[cellLinearIndex];
    if (linearThreadIndex == 0)
        *plp.f->numActiveCells = 0;
    plp.s->perCellNumAccesses[cellLinearIndex] = 0;
    if (frameIndex - lastAccessFrameIndex > 8)
        return;

    //uint32_t lightSlotIndex = linearThreadIndex % kNumLightSlotsPerCell;
    uint32_t iz = cellLinearIndex / (plp.s->gridDimension.x * plp.s->gridDimension.y);
    uint32_t iy = (cellLinearIndex % (plp.s->gridDimension.x * plp.s->gridDimension.y)) / plp.s->gridDimension.x;
    uint32_t ix = cellLinearIndex % plp.s->gridDimension.x;
    float3 cellCenter = plp.s->gridOrigin + make_float3(
        (ix + 0.5f) * plp.s->gridCellSize.x,
        (iy + 0.5f) * plp.s->gridCellSize.y,
        (iz + 0.5f) * plp.s->gridCellSize.z);
    const float minSquaredDistance = sqLength(0.5f * plp.s->gridCellSize);

    uint32_t bufferIndex = plp.f->bufferIndex;
    Reservoir<LightSample>* curReservoirs = plp.s->reservoirs[bufferIndex];
    ReservoirInfo* curReservoirInfos = plp.s->reservoirInfos[bufferIndex];

    PCG32RNG rng = plp.s->lightSlotRngs[linearThreadIndex];

    float selectedTargetPDensity = 0.0f;
    Reservoir<LightSample> reservoir;
    reservoir.initialize();

    // JP: セルの代表点に到達する光度をターゲットPDFとしてStreaming RISを実行。
    // EN: Perform streaming RIS with luminous intensity reaching to a cell's representative point
    //     as the target PDF.
    const uint32_t numCandidates = 1 << plp.f->log2NumCandidatesPerLightSlot;
    for (int candIdx = 0; candIdx < numCandidates; ++candIdx) {
        // JP: 環境光テクスチャーが設定されている場合は一定の確率でサンプルする。
        //     ダイバージェンスを抑えるために、ループの最初とそれ以外で環境光かそれ以外のサンプリングを分ける。
        //     ただし、そもそもReGIRは2段階のRISにおいてVisibilityを一切考慮していないため、環境光は(特に高いエネルギーの場合)、
        //     Reservoir中のサンプルに無駄なものを増やしてしまい、むしろ分散が増える傾向にある。
        //     環境光のサンプリングは別で行うほうが良いかもしれない。
        // EN: Sample an environmental light texture with a fixed probability if it is set.
        //     Separate sampling from the environmental light and the others to
        //     the beginning of the loop and the rest to avoid divergence.
        //     However in the first place, ReGIR doesn't take visibility into account at all during two-stage RIS,
        //     therefore an environmental light (particularly with a high-energy case) tends to increase useless
        //     samples in reservoirs, resulting in high variance.
        //     Separated environmental light sampling may be preferred.
        float uLight = rng.getFloat0cTo1o();
        bool sampleEnvLight = false;
        float probToSampleCurLightType = 1.0f;
        if (plp.s->envLightTexture && plp.f->enableEnvLight) {
            float prob = min(max(probToSampleEnvLight * numCandidates - candIdx, 0.0f), 1.0f);
            if (uLight < prob) {
                probToSampleCurLightType = probToSampleEnvLight;
                uLight = uLight / prob;
                sampleEnvLight = true;
            }
            else {
                probToSampleCurLightType = 1.0f - probToSampleEnvLight;
                uLight = (uLight - prob) / (1 - prob);
            }
        }

        // JP: 候補サンプルを生成して、ターゲットPDFを計算する。
        //     ターゲットPDFは正規化されていなくても良い。
        // EN: Generate a candidate sample then calculate the target PDF for it.
        //     Target PDF doesn't require to be normalized.
        LightSample lightSample;
        float areaPDensity;
        float3 cont = sampleIntensity(
            cellCenter, minSquaredDistance,
            uLight, sampleEnvLight, rng.getFloat0cTo1o(), rng.getFloat0cTo1o(),
            &lightSample, &areaPDensity);
        areaPDensity *= probToSampleCurLightType;
        float targetPDensity = convertToWeight(cont);

        // JP: 候補サンプル生成用のPDFとターゲットPDFは異なるためサンプルにはウェイトがかかる。
        // EN: The sample has a weight since the PDF to generate the candidate sample and the target PDF are
        //     different.
        float weight = targetPDensity / areaPDensity;
        //if (ix == 7 && iz == 7 && iy == 0) {
        //    printf("%2u, %2u, %2u, %3u, %u: %g, %g\n", ix, iy, iz, lightSlotIndex, candIdx,
        //           areaPDensity, targetPDensity);
        //}
        if (reservoir.update(lightSample, weight, rng.getFloat0cTo1o()))
            selectedTargetPDensity = targetPDensity;
    }

    // JP: 現在のサンプルが生き残る確率密度の逆数の推定値を計算する。
    // EN: Calculate the estimate of the reciprocal of the probability density that the current sample suvives.
    float recPDFEstimate = reservoir.getSumWeights() / (selectedTargetPDensity * reservoir.getStreamLength());
    if (!isfinite(recPDFEstimate)) {
        recPDFEstimate = 0.0f;
        selectedTargetPDensity = 0.0f;
    }

    // JP: 元の文献では過去数フレーム分のストリーム長で正規化されたReservoirを保持して、それらを結合しているが、
    //     ここでは正規化は行わず現在フレームと過去フレームの累積Reservoirの2つを結合する。
    // EN: The original literature suggests using stream length normalized reservoirs of several previous
    //     frames, then combine them, but here it doesn't use normalization and combines two reservoirs, one from
    //     the current frame and the other is the accumulation of the previous frames.
    if constexpr (useTemporalReuse) {
        uint32_t prevBufferIndex = (bufferIndex + 1) % 2;
        const Reservoir<LightSample>* prevReservoirs = plp.s->reservoirs[prevBufferIndex];
        const ReservoirInfo* prevReservoirInfos = plp.s->reservoirInfos[prevBufferIndex];

        uint32_t selfStreamLength = reservoir.getStreamLength();
        if (recPDFEstimate == 0.0f)
            reservoir.initialize();
        uint32_t combinedStreamLength = selfStreamLength;
        uint32_t maxNumPrevSamples = 20 * selfStreamLength;

        // JP: 際限なく過去フレームで得たサンプルがウェイトを増やさないように、
        //     前フレームのストリーム長を、現在フレームのReservoirに対して20倍までに制限する。
        // EN: Limit the stream length of the previous frame by 20 times of that of the current frame
        //     in order to avoid a sample obtained in the past getting a unlimited weight.
        // TODO: 光源アニメーションがある場合には前フレームと今のフレームでターゲットPDFが異なるので
        //       ウェイトを調整するべき？
        const Reservoir<LightSample> &prevReservoir = prevReservoirs[linearThreadIndex];
        const ReservoirInfo &prevResInfo = prevReservoirInfos[linearThreadIndex];
        const LightSample &prevLightSample = prevReservoir.getSample();
        float prevTargetDensity = prevResInfo.targetDensity;
        uint32_t prevStreamLength = min(prevReservoir.getStreamLength(), maxNumPrevSamples);
        float lengthCorrection = static_cast<float>(prevStreamLength) / prevReservoir.getStreamLength();
        float weight = lengthCorrection * prevReservoir.getSumWeights(); // New target PDF and prev target PDF are the same here.
        if (reservoir.update(prevLightSample, weight, rng.getFloat0cTo1o()))
            selectedTargetPDensity = prevTargetDensity;
        combinedStreamLength += prevStreamLength;
        reservoir.setStreamLength(combinedStreamLength);

        // JP: 現在のサンプルが生き残る確率密度の逆数の推定値を計算する。
        // EN: Calculate the estimate of the reciprocal of the probability density that the current sample suvives.
        float weightForEstimate = 1.0f / reservoir.getStreamLength();
        recPDFEstimate = weightForEstimate * reservoir.getSumWeights() / selectedTargetPDensity;
        if (!isfinite(recPDFEstimate)) {
            recPDFEstimate = 0.0f;
            selectedTargetPDensity = 0.0f;
        }
    }

    ReservoirInfo resInfo;
    resInfo.recPDFEstimate = recPDFEstimate;
    resInfo.targetDensity = selectedTargetPDensity;

    plp.s->lightSlotRngs[linearThreadIndex] = rng;
    curReservoirs[linearThreadIndex] = reservoir;
    curReservoirInfos[linearThreadIndex] = resInfo;
}

CUDA_DEVICE_KERNEL void buildCellReservoirs(PipelineLaunchParameters _plp, uint32_t frameIndex) {
    buildCellReservoirsAndTemporalReuse<false>(_plp, frameIndex);
}

CUDA_DEVICE_KERNEL void buildCellReservoirsAndTemporalReuse(PipelineLaunchParameters _plp, uint32_t frameIndex) {
    buildCellReservoirsAndTemporalReuse<true>(_plp, frameIndex);
}

CUDA_DEVICE_KERNEL void updateLastAccessFrameIndices(PipelineLaunchParameters _plp, uint32_t frameIndex) {
    plp = _plp;

    // JP: 現在のフレーム中でアクセスされたセルにフレーム番号を記録する。
    // EN: Record the frame number to cells that accessed in the current frame.
    uint32_t linearThreadIndex = blockDim.x * blockIdx.x + threadIdx.x;
    uint32_t cellLinearIndex = linearThreadIndex;
    uint32_t perCellNumAccesses = plp.s->perCellNumAccesses[cellLinearIndex];
    if (perCellNumAccesses > 0)
        plp.s->lastAccessFrameIndices[cellLinearIndex] = frameIndex;

    uint32_t numActiveCellsInGroup = __popc(__ballot_sync(0xFFFFFFFF, perCellNumAccesses > 0));
    if (threadIdx.x == 0 && numActiveCellsInGroup > 0)
        atomicAdd(plp.f->numActiveCells, numActiveCellsInGroup);
}
