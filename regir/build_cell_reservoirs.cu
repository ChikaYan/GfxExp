#include "hip/hip_runtime.h"
#include "regir_shared.h"
#include "../common/common_device.cuh"

using namespace shared;

CUDA_DEVICE_MEM static PipelineLaunchParameters plp;

CUDA_DEVICE_FUNCTION float3 sampleLight(
    float ul, bool sampleEnvLight, float u0, float u1,
    LightSample* lightSample, float3* lightPosition, float3* lightNormal, float* areaPDensity) {
    hipTextureObject_t texEmittance = 0;
    float3 emittance = make_float3(0.0f, 0.0f, 0.0f);
    float2 texCoord;
    if (sampleEnvLight) {
        lightSample->instIndex = 0xFFFFFFFF;
        lightSample->geomInstIndex = 0xFFFFFFFF;
        lightSample->primIndex = 0xFFFFFFFF;

        float u, v;
        float uvPDF;
        plp.s->envLightImportanceMap.sample(u0, u1, &u, &v, &uvPDF);
        float phi = 2 * Pi * u;
        float theta = Pi * v;
        lightSample->b1 = phi;
        lightSample->b2 = theta;

        float posPhi = phi - plp.f->envLightRotation;
        posPhi = posPhi - floorf(posPhi / (2 * Pi)) * 2 * Pi;

        float3 direction = fromPolarYUp(posPhi, theta);
        float3 position = make_float3(direction.x, direction.y, direction.z);
        *lightPosition = position;

        *lightNormal = -position;

        // JP: �e�N�X�`���[��Ԓ���PDF��ʐςɊւ�����̂ɕϊ�����B
        // EN: convert the PDF in texture space to one with respect to area.
        // The true value is: lim_{l to inf} uvPDF / (2 * Pi * Pi * std::sin(theta)) / l^2
        *areaPDensity = uvPDF / (2 * Pi * Pi * std::sin(theta));

        texEmittance = plp.s->envLightTexture;
        // JP: ���}�b�v�e�N�X�`���[�̒l�ɌW���������āA�ʏ�̌����Ɠ����悤�ɕԂ�l���������U�x
        //     �Ƃ��Ĉ�����悤�ɂ���B
        // EN: Multiply a coefficient to make the return value possible to be handled as luminous emittance.
        emittance = make_float3(Pi * plp.f->envLightPowerCoeff);
        texCoord.x = u;
        texCoord.y = v;
    }
    else {
        float lightProb = 1.0f;

        // JP: �܂��̓C���X�^���X���T���v������B
        // EN: First, sample an instance.
        float instProb;
        float uGeomInst;
        uint32_t instIndex = plp.s->lightInstDist.sample(ul, &instProb, &uGeomInst);
        lightProb *= instProb;
        const InstanceData &inst = plp.f->instanceDataBuffer[instIndex];
        lightSample->instIndex = instIndex;

        // JP: ���ɃT���v�������C���X�^���X�ɑ�����W�I���g���C���X�^���X���T���v������B
        // EN: Next, sample a geometry instance which belongs to the sampled instance.
        float geomInstProb;
        float uPrim;
        uint32_t geomInstIndexInInst = inst.lightGeomInstDist.sample(uGeomInst, &geomInstProb, &uPrim);
        uint32_t geomInstIndex = inst.geomInstSlots[geomInstIndexInInst];
        lightProb *= geomInstProb;
        const GeometryInstanceData &geomInst = plp.s->geometryInstanceDataBuffer[geomInstIndex];
        lightSample->geomInstIndex = geomInstIndex;

        // JP: �Ō�ɁA�T���v�������W�I���g���C���X�^���X�ɑ�����v���~�e�B�u���T���v������B
        // EN: Finally, sample a primitive which belongs to the sampled geometry instance.
        float primProb;
        uint32_t primIndex = geomInst.emitterPrimDist.sample(uPrim, &primProb);
        lightProb *= primProb;
        lightSample->primIndex = primIndex;

        // Uniform sampling on unit triangle
        // A Low-Distortion Map Between Triangle and Square
        float t0 = 0.5f * u0;
        float t1 = 0.5f * u1;
        float offset = t1 - t0;
        if (offset > 0)
            t1 += offset;
        else
            t0 -= offset;
        float t2 = 1 - (t0 + t1);

        lightSample->b1 = t1;
        lightSample->b2 = t2;

        //printf("%u-%u-%u: %g\n", instIndex, geomInstIndex, primIndex, lightProb);

        const MaterialData &mat = plp.s->materialDataBuffer[geomInst.materialSlot];

        const shared::Triangle &tri = geomInst.triangleBuffer[primIndex];
        const shared::Vertex (&v)[3] = {
            geomInst.vertexBuffer[tri.index0],
            geomInst.vertexBuffer[tri.index1],
            geomInst.vertexBuffer[tri.index2]
        };
        float3 p[3] = {
            inst.transform * v[0].position,
            inst.transform * v[1].position,
            inst.transform * v[2].position,
        };

        float3 geomNormal = cross(p[1] - p[0], p[2] - p[0]);
        *lightPosition = t0 * p[0] + t1 * p[1] + t2 * p[2];
        float recArea = 1.0f / length(geomNormal);
        //*lightNormal = geomNormal * recArea;
        *lightNormal = t0 * v[0].normal + t1 * v[1].normal + t2 * v[2].normal;
        *lightNormal = normalize(inst.normalMatrix * *lightNormal);
        recArea *= 2;
        *areaPDensity = lightProb * recArea;

        //printf("%u-%u-%u: (%g, %g, %g), PDF: %g\n", instIndex, geomInstIndex, primIndex,
        //       mat.emittance.x, mat.emittance.y, mat.emittance.z, *areaPDensity);

        //printf("%u-%u-%u: (%g, %g, %g), (%g, %g, %g)\n", instIndex, geomInstIndex, primIndex,
        //       lightPosition->x, lightPosition->y, lightPosition->z,
        //       lightNormal->x, lightNormal->y, lightNormal->z);

        if (mat.emittance) {
            texEmittance = mat.emittance;
            emittance = make_float3(1.0f, 1.0f, 1.0f);
            texCoord = t0 * v[0].texCoord + t1 * v[1].texCoord + t2 * v[2].texCoord;
        }
    }

    if (texEmittance) {
        float4 texValue = tex2DLod<float4>(texEmittance, texCoord.x, texCoord.y, 0.0f);
        emittance *= make_float3(texValue);
    }

    return emittance;
}

// TODO: �Z���̒��������̃T���v�����O���ƁA�Z���̒����������̗����ɉ���Ă��܂��Ă���ꍇ�ɁA
//       ��^�̉\���̂���T���v�������p���Ă��܂��B��\�_�������_���Ɍ��肷��Ȃǂŉ����ł����������A
//       PDF������ς��̂ł�����l������K�v����H
CUDA_DEVICE_FUNCTION float3 sampleIntensity(
    const float3 &shadingPoint, float minSquaredDistance,
    float uLight, bool sampleEnvLight, float uPos0, float uPos1,
    LightSample* lightSample, float* probDensity) {
    float3 lp;
    float3 lpn;
    float3 M = sampleLight(uLight, sampleEnvLight, uPos0, uPos1,
                           lightSample, &lp, &lpn, probDensity);
    bool atInfinity = lightSample->atInfinity();

    float3 shadowRayDir = atInfinity ? lp : (lp - shadingPoint);
    float dist2 = sqLength(shadowRayDir);
    float dist = std::sqrt(dist2);
    shadowRayDir /= dist;

    float lpCos = dot(-shadowRayDir, lpn);

    if (lpCos > 0) {
        float3 Le = M / Pi;
        float3 ret = Le * (lpCos / dist2);
        return ret;
    }
    else {
        return make_float3(0.0f, 0.0f, 0.0f);
    }
}

template <bool useTemporalReuse>
CUDA_DEVICE_FUNCTION void buildCellReservoirsAndTemporalReuse(const PipelineLaunchParameters &_plp, uint32_t frameIndex) {
    plp = _plp;

    uint32_t linearThreadIndex = blockDim.x * blockIdx.x + threadIdx.x;
    uint32_t cellLinearIndex = linearThreadIndex / kNumLightSlotsPerCell;
    uint32_t lastAccessFrameIndex = plp.s->lastAccessFrameIndices[cellLinearIndex];
    if (linearThreadIndex == 0)
        *plp.f->numActiveCells = 0;
    plp.s->perCellNumAccesses[cellLinearIndex] = 0;
    if (frameIndex - lastAccessFrameIndex > 8)
        return;

    //uint32_t lightSlotIndex = linearThreadIndex % kNumLightSlotsPerCell;
    uint32_t iz = cellLinearIndex / (plp.s->gridDimension.x * plp.s->gridDimension.y);
    uint32_t iy = (cellLinearIndex % (plp.s->gridDimension.x * plp.s->gridDimension.y)) / plp.s->gridDimension.x;
    uint32_t ix = cellLinearIndex % plp.s->gridDimension.x;
    float3 cellCenter = plp.s->gridOrigin + make_float3(
        (ix + 0.5f) * plp.s->gridCellSize.x,
        (iy + 0.5f) * plp.s->gridCellSize.y,
        (iz + 0.5f) * plp.s->gridCellSize.z);
    const float minSquaredDistance = sqLength(0.5f * plp.s->gridCellSize);

    uint32_t bufferIndex = plp.f->bufferIndex;
    Reservoir<LightSample>* curReservoirs = plp.s->reservoirs[bufferIndex];
    ReservoirInfo* curReservoirInfos = plp.s->reservoirInfos[bufferIndex];

    PCG32RNG rng = plp.s->lightSlotRngs[linearThreadIndex];

    float selectedTargetPDensity = 0.0f;
    Reservoir<LightSample> reservoir;
    reservoir.initialize();

    // JP: �Z���̑�\�_�ɓ��B������x���^�[�Q�b�gPDF�Ƃ���Streaming RIS�����s�B
    // EN: Perform streaming RIS with luminous intensity reaching to a cell's representative point
    //     as the target PDF.
    const uint32_t numCandidates = 1 << plp.f->log2NumCandidatesPerLightSlot;
    for (int candIdx = 0; candIdx < numCandidates; ++candIdx) {
        // JP: �����e�N�X�`���[���ݒ肳��Ă���ꍇ�͈��̊m���ŃT���v������B
        //     �_�C�o�[�W�F���X��}���邽�߂ɁA���[�v�̍ŏ��Ƃ���ȊO�Ŋ���������ȊO�̃T���v�����O�𕪂���B
        //     �������A��������ReGIR��2�i�K��RIS�ɂ�����Visibility����؍l�����Ă��Ȃ����߁A������(���ɍ����G�l���M�[�̏ꍇ)�A
        //     Reservoir���̃T���v���ɖ��ʂȂ��̂𑝂₵�Ă��܂��A�ނ��땪�U��������X���ɂ���B
        //     �����̃T���v�����O�͕ʂōs���ق����ǂ���������Ȃ��B
        // EN: Sample an environmental light texture with a fixed probability if it is set.
        //     Separate sampling from the environmental light and the others to
        //     the beginning of the loop and the rest to avoid divergence.
        //     However in the first place, ReGIR doesn't take visibility into account at all during two-stage RIS,
        //     therefore an environmental light (particularly with a high-energy case) tends to increase useless
        //     samples in reservoirs, resulting in high variance.
        //     Separated environmental light sampling may be preferred.
        float uLight = rng.getFloat0cTo1o();
        bool sampleEnvLight = false;
        float probToSampleCurLightType = 1.0f;
        if (plp.s->envLightTexture && plp.f->enableEnvLight) {
            float prob = min(max(probToSampleEnvLight * numCandidates - candIdx, 0.0f), 1.0f);
            if (uLight < prob) {
                probToSampleCurLightType = probToSampleEnvLight;
                uLight = uLight / prob;
                sampleEnvLight = true;
            }
            else {
                probToSampleCurLightType = 1.0f - probToSampleEnvLight;
                uLight = (uLight - prob) / (1 - prob);
            }
        }

        // JP: ���T���v���𐶐����āA�^�[�Q�b�gPDF���v�Z����B
        //     �^�[�Q�b�gPDF�͐��K������Ă��Ȃ��Ă��ǂ��B
        // EN: Generate a candidate sample then calculate the target PDF for it.
        //     Target PDF doesn't require to be normalized.
        LightSample lightSample;
        float areaPDensity;
        float3 cont = sampleIntensity(
            cellCenter, minSquaredDistance,
            uLight, sampleEnvLight, rng.getFloat0cTo1o(), rng.getFloat0cTo1o(),
            &lightSample, &areaPDensity);
        areaPDensity *= probToSampleCurLightType;
        float targetPDensity = convertToWeight(cont);

        // JP: ���T���v�������p��PDF�ƃ^�[�Q�b�gPDF�͈قȂ邽�߃T���v���ɂ̓E�F�C�g��������B
        // EN: The sample has a weight since the PDF to generate the candidate sample and the target PDF are
        //     different.
        float weight = targetPDensity / areaPDensity;
        //if (ix == 7 && iz == 7 && iy == 0) {
        //    printf("%2u, %2u, %2u, %3u, %u: %g, %g\n", ix, iy, iz, lightSlotIndex, candIdx,
        //           areaPDensity, targetPDensity);
        //}
        if (reservoir.update(lightSample, weight, rng.getFloat0cTo1o()))
            selectedTargetPDensity = targetPDensity;
    }

    // JP: ���݂̃T���v���������c��m�����x�̋t���̐���l���v�Z����B
    // EN: Calculate the estimate of the reciprocal of the probability density that the current sample suvives.
    float recPDFEstimate = reservoir.getSumWeights() / (selectedTargetPDensity * reservoir.getStreamLength());
    if (!isfinite(recPDFEstimate)) {
        recPDFEstimate = 0.0f;
        selectedTargetPDensity = 0.0f;
    }

    // JP: ���̕����ł͉ߋ����t���[�����̃X�g���[�����Ő��K�����ꂽReservoir��ێ����āA�������������Ă��邪�A
    //     �����ł͐��K���͍s�킸���݃t���[���Ɖߋ��t���[���̗ݐ�Reservoir��2����������B
    // EN: The original literature suggests using stream length normalized reservoirs of several previous
    //     frames, then combine them, but here it doesn't use normalization and combines two reservoirs, one from
    //     the current frame and the other is the accumulation of the previous frames.
    if constexpr (useTemporalReuse) {
        uint32_t prevBufferIndex = (bufferIndex + 1) % 2;
        const Reservoir<LightSample>* prevReservoirs = plp.s->reservoirs[prevBufferIndex];
        const ReservoirInfo* prevReservoirInfos = plp.s->reservoirInfos[prevBufferIndex];

        uint32_t selfStreamLength = reservoir.getStreamLength();
        if (recPDFEstimate == 0.0f)
            reservoir.initialize();
        uint32_t combinedStreamLength = selfStreamLength;
        uint32_t maxNumPrevSamples = 20 * selfStreamLength;

        // JP: �ی��Ȃ��ߋ��t���[���œ����T���v�����E�F�C�g�𑝂₳�Ȃ��悤�ɁA
        //     �O�t���[���̃X�g���[�������A���݃t���[����Reservoir�ɑ΂���20�{�܂łɐ�������B
        // EN: Limit the stream length of the previous frame by 20 times of that of the current frame
        //     in order to avoid a sample obtained in the past getting a unlimited weight.
        // TODO: �����A�j���[�V����������ꍇ�ɂ͑O�t���[���ƍ��̃t���[���Ń^�[�Q�b�gPDF���قȂ�̂�
        //       �E�F�C�g�𒲐�����ׂ��H
        const Reservoir<LightSample> &prevReservoir = prevReservoirs[linearThreadIndex];
        const ReservoirInfo &prevResInfo = prevReservoirInfos[linearThreadIndex];
        const LightSample &prevLightSample = prevReservoir.getSample();
        float prevTargetDensity = prevResInfo.targetDensity;
        uint32_t prevStreamLength = min(prevReservoir.getStreamLength(), maxNumPrevSamples);
        float lengthCorrection = static_cast<float>(prevStreamLength) / prevReservoir.getStreamLength();
        float weight = lengthCorrection * prevReservoir.getSumWeights(); // New target PDF and prev target PDF are the same here.
        if (reservoir.update(prevLightSample, weight, rng.getFloat0cTo1o()))
            selectedTargetPDensity = prevTargetDensity;
        combinedStreamLength += prevStreamLength;
        reservoir.setStreamLength(combinedStreamLength);

        // JP: ���݂̃T���v���������c��m�����x�̋t���̐���l���v�Z����B
        // EN: Calculate the estimate of the reciprocal of the probability density that the current sample suvives.
        float weightForEstimate = 1.0f / reservoir.getStreamLength();
        recPDFEstimate = weightForEstimate * reservoir.getSumWeights() / selectedTargetPDensity;
        if (!isfinite(recPDFEstimate)) {
            recPDFEstimate = 0.0f;
            selectedTargetPDensity = 0.0f;
        }
    }

    ReservoirInfo resInfo;
    resInfo.recPDFEstimate = recPDFEstimate;
    resInfo.targetDensity = selectedTargetPDensity;

    plp.s->lightSlotRngs[linearThreadIndex] = rng;
    curReservoirs[linearThreadIndex] = reservoir;
    curReservoirInfos[linearThreadIndex] = resInfo;
}

CUDA_DEVICE_KERNEL void buildCellReservoirs(PipelineLaunchParameters _plp, uint32_t frameIndex) {
    buildCellReservoirsAndTemporalReuse<false>(_plp, frameIndex);
}

CUDA_DEVICE_KERNEL void buildCellReservoirsAndTemporalReuse(PipelineLaunchParameters _plp, uint32_t frameIndex) {
    buildCellReservoirsAndTemporalReuse<true>(_plp, frameIndex);
}

CUDA_DEVICE_KERNEL void updateLastAccessFrameIndices(PipelineLaunchParameters _plp, uint32_t frameIndex) {
    plp = _plp;

    // JP: ���݂̃t���[�����ŃA�N�Z�X���ꂽ�Z���Ƀt���[���ԍ����L�^����B
    // EN: Record the frame number to cells that accessed in the current frame.
    uint32_t linearThreadIndex = blockDim.x * blockIdx.x + threadIdx.x;
    uint32_t cellLinearIndex = linearThreadIndex;
    uint32_t perCellNumAccesses = plp.s->perCellNumAccesses[cellLinearIndex];
    if (perCellNumAccesses > 0)
        plp.s->lastAccessFrameIndices[cellLinearIndex] = frameIndex;

    uint32_t numActiveCellsInGroup = __popc(__ballot_sync(0xFFFFFFFF, perCellNumAccesses > 0));
    if (threadIdx.x == 0 && numActiveCellsInGroup > 0)
        atomicAdd(plp.f->numActiveCells, numActiveCellsInGroup);
}
