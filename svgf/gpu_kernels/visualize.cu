#include "hip/hip_runtime.h"
#define PURE_CUDA
#include "../svgf_shared.h"

using namespace shared;

CUDA_DEVICE_KERNEL void debugVisualize(
    BufferToDisplay bufferToDisplay,
    float motionVectorOffset, float motionVectorScale) {
    uint2 launchIndex = make_uint2(blockDim.x * blockIdx.x + threadIdx.x,
                                   blockDim.y * blockIdx.y + threadIdx.y);
    int2 pix = make_int2(launchIndex.x, launchIndex.y);
    int2 imageSize = plp.s->imageSize;
    if (launchIndex.x >= imageSize.x ||
        launchIndex.y >= imageSize.y)
        return;

    uint32_t curBufIdx = plp.f->bufferIndex;
    const StaticPipelineLaunchParameters::TemporalSet &staticTemporalSet =
        plp.s->temporalSets[curBufIdx];
    const PerFramePipelineLaunchParameters::TemporalSet &perFrameTemporalSet =
        plp.f->temporalSets[curBufIdx];

    float4 color = make_float4(0.0f, 0.0f, 0.0f, 1.0f);
    switch (bufferToDisplay) {
    case BufferToDisplay::NoisyBeauty:
        break;
    case BufferToDisplay::Albedo: {
        Albedo albedo = plp.s->albedoBuffer.read(pix);
        color = make_float4(albedo.dhReflectance, 1.0f);
        break;
    }
    case BufferToDisplay::Normal: {
        GBuffer1 gBuffer1 = perFrameTemporalSet.GBuffer1.read(glPix(pix));
        color = make_float4(0.5f * gBuffer1.normalInWorld + make_float3(0.5f), 1.0f);
        break;
    }
    case BufferToDisplay::Flow: {
        GBuffer2 gBuffer2 = perFrameTemporalSet.GBuffer2.read(glPix(pix));
        float2 curScreenPos = make_float2(pix.x + 0.5f, pix.y + 0.5f) / imageSize;
        float2 prevScreenPos = gBuffer2.prevScreenPos;
        float2 motionVector = imageSize * (curScreenPos - prevScreenPos);
        color = make_float4(clamp(motionVectorScale * motionVector.x + motionVectorOffset, 0.0f, 1.0f),
                            clamp(motionVectorScale * motionVector.y + motionVectorOffset, 0.0f, 1.0f),
                            motionVectorOffset, 1.0f);
        break;
    }
    case BufferToDisplay::SampleCount: {
        MomentPair_SampleInfo momentPair_sampleInfo =
            staticTemporalSet.momentPair_sampleInfo_buffer.read(pix);
        float value = min(momentPair_sampleInfo.sampleInfo.count / 255.0f, 1.0f);
        color = make_float4(make_float3(value), 1.0f);
        break;
    }
    default:
        Assert_ShouldNotBeCalled();
        break;
    }

    plp.f->debugVisualizeBuffer.write(glPix(pix), color);
}
