#include "hip/hip_runtime.h"
#define PURE_CUDA
#include "restir_shared.h"

using namespace shared;

CUDA_DEVICE_KERNEL void performLightPreSampling() {
    uint32_t linearThreadIndex = blockDim.x * blockIdx.x + threadIdx.x;
    //uint32_t subsetIndex = linearThreadIndex / lightSubsetSize;
    uint32_t indexInSubset = linearThreadIndex % lightSubsetSize;
    PCG32RNG rng = plp.s->lightPreSamplingRngs[linearThreadIndex];
    float probToSampleCurLightType = 1.0f;
    bool sampleEnvLight = false;
    if (plp.s->envLightTexture && plp.f->enableEnvLight) {
        sampleEnvLight = indexInSubset < probToSampleEnvLight * lightSubsetSize;
        probToSampleCurLightType = sampleEnvLight ?
            probToSampleEnvLight : (1 - probToSampleEnvLight);
    }
    PreSampledLight preSampledLight;
    sampleLight(
        rng.getFloat0cTo1o(), sampleEnvLight, rng.getFloat0cTo1o(), rng.getFloat0cTo1o(),
        &preSampledLight.sample, &preSampledLight.areaPDensity);
    preSampledLight.areaPDensity *= probToSampleCurLightType;

    plp.s->lightPreSamplingRngs[linearThreadIndex] = rng;
    plp.s->preSampledLights[linearThreadIndex] = preSampledLight;
}



CUDA_DEVICE_KERNEL void performPerPixelRIS() {
    int2 launchIndex = make_int2(blockDim.x * blockIdx.x + threadIdx.x,
                                 blockDim.y * blockIdx.y + threadIdx.y);

    uint32_t curBufIdx = plp.f->bufferIndex;
    GBuffer0 gBuffer0 = plp.s->GBuffer0[curBufIdx].read(launchIndex);
    GBuffer1 gBuffer1 = plp.s->GBuffer1[curBufIdx].read(launchIndex);
    GBuffer2 gBuffer2 = plp.s->GBuffer2[curBufIdx].read(launchIndex);

    float3 positionInWorld = gBuffer0.positionInWorld;
    float3 shadingNormalInWorld = gBuffer1.normalInWorld;
    float2 texCoord = make_float2(gBuffer0.texCoord_x, gBuffer1.texCoord_y);
    uint32_t materialSlot = gBuffer2.materialSlot;

    PCG32RNG rng = plp.s->rngBuffer.read(launchIndex);
    CUDA_SHARED_MEM uint32_t sm_perTileLightSubsetIndex;
    if (threadIdx.x == 0 && threadIdx.y == 0)
        sm_perTileLightSubsetIndex = mapPrimarySampleToDiscrete(rng.getFloat0cTo1o(), numLightSubsets);
    __syncthreads();
    uint32_t perTileLightSubsetIndex = sm_perTileLightSubsetIndex;
    const PreSampledLight* lightSubSet = &plp.s->preSampledLights[perTileLightSubsetIndex * lightSubsetSize];

    if (materialSlot == 0xFFFFFFFF)
        return;

    const MaterialData &mat = plp.s->materialDataBuffer[materialSlot];

    // TODO?: Use true geometric normal.
    float3 geometricNormalInWorld = shadingNormalInWorld;
    float3 vOut = plp.f->camera.position - positionInWorld;
    float frontHit = dot(vOut, geometricNormalInWorld) >= 0.0f ? 1.0f : -1.0f;

    BSDF bsdf;
    mat.setupBSDF(mat, texCoord, &bsdf);
    ReferenceFrame shadingFrame(shadingNormalInWorld);
    positionInWorld = offsetRayOriginNaive(positionInWorld, frontHit * geometricNormalInWorld);
    float dist = length(vOut);
    vOut /= dist;
    float3 vOutLocal = shadingFrame.toLocal(vOut);

    uint32_t curResIndex = plp.currentReservoirIndex;
    Reservoir<LightSample> reservoir;
    reservoir.initialize();

    // JP: Unshadowed Contribution���^�[�Q�b�gPDF�Ƃ���Streaming RIS�����s�B
    // EN: Perform streaming RIS with unshadowed contribution as the target PDF.
    float selectedTargetDensity = 0.0f;
    uint32_t numCandidates = 1 << plp.f->log2NumCandidateSamples;
    for (int i = 0; i < numCandidates; ++i) {
        uint32_t lightIndex = mapPrimarySampleToDiscrete(rng.getFloat0cTo1o(), lightSubsetSize);
        const PreSampledLight &preSampledLight = lightSubSet[lightIndex];

        // JP: ���T���v���𐶐����āA�^�[�Q�b�gPDF���v�Z����B
        //     �^�[�Q�b�gPDF�͐��K������Ă��Ȃ��Ă��ǂ��B
        // EN: Generate a candidate sample then calculate the target PDF for it.
        //     Target PDF doesn't require to be normalized.
        float3 cont = performDirectLighting<false>(
            positionInWorld, vOutLocal, shadingFrame, bsdf,
            preSampledLight.sample);
        float targetDensity = convertToWeight(cont);

        // JP: ���T���v�������p��PDF�ƃ^�[�Q�b�gPDF�͈قȂ邽�߃T���v���ɂ̓E�F�C�g��������B
        // EN: The sample has a weight since the PDF to generate the candidate sample and the target PDF are
        //     different.
        float weight = targetDensity / preSampledLight.areaPDensity;
        if (reservoir.update(preSampledLight.sample, weight, rng.getFloat0cTo1o()))
            selectedTargetDensity = targetDensity;
    }

    // JP: ���݂̃T���v���������c��m�����x�̋t���̐���l���v�Z����B
    // EN: Calculate the estimate of the reciprocal of the probability density that the current sample suvives.
    float recPDFEstimate = reservoir.getSumWeights() / (selectedTargetDensity * reservoir.getStreamLength());
    if (!isfinite(recPDFEstimate)) {
        recPDFEstimate = 0.0f;
        selectedTargetDensity = 0.0f;
    }

    ReservoirInfo reservoirInfo;
    reservoirInfo.recPDFEstimate = recPDFEstimate;
    reservoirInfo.targetDensity = selectedTargetDensity;

    plp.s->rngBuffer.write(launchIndex, rng);
    plp.s->reservoirBuffer[curResIndex][launchIndex] = reservoir;
    plp.s->reservoirInfoBuffer[curResIndex].write(launchIndex, reservoirInfo);
}
