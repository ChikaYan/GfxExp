#include "hip/hip_runtime.h"
#define PURE_CUDA
#include "restir_shared.h"

using namespace shared;

CUDA_DEVICE_KERNEL void performLightPreSampling() {
    uint32_t linearThreadIndex = blockDim.x * blockIdx.x + threadIdx.x;
    //uint32_t subsetIndex = linearThreadIndex / lightSubsetSize;
    uint32_t indexInSubset = linearThreadIndex % lightSubsetSize;
    PCG32RNG rng = plp.s->lightPreSamplingRngs[linearThreadIndex];

    // JP: �����e�N�X�`���[���ݒ肳��Ă���ꍇ�͈��̊m���ŃT���v������B
    //     �_�C�o�[�W�F���X��}���邽�߂ɁA�T�u�Z�b�g�̍ŏ��Ƃ���ȊO�Ŋ���������ȊO�̃T���v�����O�𕪂���B
    // EN: Sample an environmental light texture with a fixed probability if it is set.
    //     Separate sampling from the environmental light and the others to
    //     the beginning of the subset and the rest to avoid divergence.
    float probToSampleCurLightType = 1.0f;
    bool sampleEnvLight = false;
    if (plp.s->envLightTexture && plp.f->enableEnvLight) {
        if (plp.s->lightInstDist.integral() > 0.0f) {
            sampleEnvLight = indexInSubset < probToSampleEnvLight * lightSubsetSize;
            probToSampleCurLightType = sampleEnvLight ?
                probToSampleEnvLight : (1 - probToSampleEnvLight);
        }
        else {
            sampleEnvLight = true;
        }
    }

    PreSampledLight preSampledLight;
    sampleLight<false>(
        make_float3(0.0f),
        rng.getFloat0cTo1o(), sampleEnvLight, rng.getFloat0cTo1o(), rng.getFloat0cTo1o(),
        &preSampledLight.sample, &preSampledLight.areaPDensity);
    preSampledLight.areaPDensity *= probToSampleCurLightType;

    plp.s->lightPreSamplingRngs[linearThreadIndex] = rng;
    plp.s->preSampledLights[linearThreadIndex] = preSampledLight;
}



CUDA_DEVICE_KERNEL void performPerPixelRIS() {
    int2 launchIndex = make_int2(blockDim.x * blockIdx.x + threadIdx.x,
                                 blockDim.y * blockIdx.y + threadIdx.y);

    uint32_t curBufIdx = plp.f->bufferIndex;
    GBuffer0 gBuffer0 = plp.s->GBuffer0[curBufIdx].read(launchIndex);
    GBuffer1 gBuffer1 = plp.s->GBuffer1[curBufIdx].read(launchIndex);
    GBuffer2 gBuffer2 = plp.s->GBuffer2[curBufIdx].read(launchIndex);

    float3 positionInWorld = gBuffer0.positionInWorld;
    float3 shadingNormalInWorld = gBuffer1.normalInWorld;
    float2 texCoord = make_float2(gBuffer0.texCoord_x, gBuffer1.texCoord_y);
    uint32_t materialSlot = gBuffer2.materialSlot;

    // JP: �^�C�����Ƃɋ��ʂ̃��C�g�T�u�Z�b�g��I�����邱�ƂŃ������A�N�Z�X�̃R�q�[�����V�[�����P����B
    // EN: Select a common light subset for each tile to improve memory access coherency.
    PCG32RNG rng = plp.s->rngBuffer.read(launchIndex);
    CUDA_SHARED_MEM uint32_t sm_perTileLightSubsetIndex;
    if (threadIdx.x == 0 && threadIdx.y == 0)
        sm_perTileLightSubsetIndex = mapPrimarySampleToDiscrete(rng.getFloat0cTo1o(), numLightSubsets);
    __syncthreads();
    uint32_t perTileLightSubsetIndex = sm_perTileLightSubsetIndex;
    const PreSampledLight* lightSubSet = &plp.s->preSampledLights[perTileLightSubsetIndex * lightSubsetSize];

    if (materialSlot == 0xFFFFFFFF)
        return;

    const MaterialData &mat = plp.s->materialDataBuffer[materialSlot];

    // TODO?: Use true geometric normal.
    float3 geometricNormalInWorld = shadingNormalInWorld;
    float3 vOut = plp.f->camera.position - positionInWorld;
    float frontHit = dot(vOut, geometricNormalInWorld) >= 0.0f ? 1.0f : -1.0f;

    BSDF bsdf;
    bsdf.setup(mat, texCoord);
    ReferenceFrame shadingFrame(shadingNormalInWorld);
    positionInWorld = offsetRayOriginNaive(positionInWorld, frontHit * geometricNormalInWorld);
    float dist = length(vOut);
    vOut /= dist;
    float3 vOutLocal = shadingFrame.toLocal(vOut);

    uint32_t curResIndex = plp.currentReservoirIndex;
    Reservoir<LightSample> reservoir;
    reservoir.initialize();

    // JP: Unshadowed Contribution���^�[�Q�b�gPDF�Ƃ���Streaming RIS�����s�B
    // EN: Perform streaming RIS with unshadowed contribution as the target PDF.
    float selectedTargetDensity = 0.0f;
    uint32_t numCandidates = 1 << plp.f->log2NumCandidateSamples;
    for (int i = 0; i < numCandidates; ++i) {
        uint32_t lightIndex = mapPrimarySampleToDiscrete(rng.getFloat0cTo1o(), lightSubsetSize);
        const PreSampledLight &preSampledLight = lightSubSet[lightIndex];

        // JP: ���T���v���𐶐����āA�^�[�Q�b�gPDF���v�Z����B
        //     �^�[�Q�b�gPDF�͐��K������Ă��Ȃ��Ă��ǂ��B
        // EN: Generate a candidate sample then calculate the target PDF for it.
        //     Target PDF doesn't require to be normalized.
        float3 cont = performDirectLighting<false>(
            positionInWorld, vOutLocal, shadingFrame, bsdf,
            preSampledLight.sample);
        float targetDensity = convertToWeight(cont);

        // JP: ���T���v�������p��PDF�ƃ^�[�Q�b�gPDF�͈قȂ邽�߃T���v���ɂ̓E�F�C�g��������B
        // EN: The sample has a weight since the PDF to generate the candidate sample and the target PDF are
        //     different.
        float weight = targetDensity / preSampledLight.areaPDensity;
        if (reservoir.update(preSampledLight.sample, weight, rng.getFloat0cTo1o()))
            selectedTargetDensity = targetDensity;
    }

    // JP: ���݂̃T���v���������c��m�����x�̋t���̐���l���v�Z����B
    // EN: Calculate the estimate of the reciprocal of the probability density that the current sample suvives.
    float recPDFEstimate = reservoir.getSumWeights() / (selectedTargetDensity * reservoir.getStreamLength());
    if (!isfinite(recPDFEstimate)) {
        recPDFEstimate = 0.0f;
        selectedTargetDensity = 0.0f;
    }

    ReservoirInfo reservoirInfo;
    reservoirInfo.recPDFEstimate = recPDFEstimate;
    reservoirInfo.targetDensity = selectedTargetDensity;

    plp.s->rngBuffer.write(launchIndex, rng);
    plp.s->reservoirBuffer[curResIndex][launchIndex] = reservoir;
    plp.s->reservoirInfoBuffer[curResIndex].write(launchIndex, reservoirInfo);
}
